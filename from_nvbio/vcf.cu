#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

// loader for variant call format files, version 4.2

#include "../bqsr_types.h"
#include "vcf.h"
#include "bufferedtextfile.h"
#include "dna.h"
#include "alphabet.h"

#include <stdlib.h>
#include <string.h>

// parse the INFO field looking for an END tag
// INFO is a set of ID=val entries separated by semicolons
// returns false if a parse error occurs
static bool get_end_position(uint32 *out, char *info)
{
    char *sc, *eq;

    do {
        // search for the next semi-colon
        sc = strchr(info, ';');
        if (sc)
        {
            // null it out
            *sc = '\0';
        }

        // now search for the next equal sign
        eq = strchr(info, '=');
        if (!eq)
        {
            // no equal sign, malformed header
            return false;
        }

        // zero out the equal sign
        *eq = 0;

        // check the key name
        if (strcmp(info, "END") == 0)
        {
            // parse the END value
            char *endptr = NULL;
            uint32 position = strtoll(eq + 1, &endptr, 10);
            if (!endptr || endptr == eq || *endptr != '\0')
            {
                return false;
            }

            *out = position;
            return true;
        }

        if (sc)
        {
            info = sc + 1;
        } else {
            info = NULL;
        }
    } while (info && *info);

    return true;
}

// loads a VCF 4.2 file, appending the data to output
bool loadVCF(SNPDatabase& output, const char *file_name)
{
    BufferedTextFile file(file_name);
    char *line, *end;
    uint32 line_counter = 0;

    while((line = file.next_record(&end)))
    {
        line_counter++;
        *end = '\0';

        // strip out comments
        char *comment = strchr(line, '#');
        if (comment)
            *comment = '\0';

        // skip all leading whitespace
        while (*line == ' ' || *line == '\t' || *line == '\r')
        {
            line++;
        }

        if (*line == '\0')
        {
            // empty line, skip
            continue;
        }

        // parse the entries in each record
        char *chrom  = NULL;
        char *pos    = NULL;
        char *id     = NULL;
        char *ref    = NULL;
        char *alt    = NULL;
        char *qual   = NULL;
        char *filter = NULL;
        char *info   = NULL;

// ugly macro to tokenize the string based on strchr
#define NEXT(prev, next)                        \
    {                                           \
        if (prev)                               \
        {                                       \
            next = strchr(prev, '\t');          \
            if (next)                           \
            {                                   \
                *next = '\0';                   \
                next++;                         \
            }                                   \
        }                                       \
    }

        chrom = line;
        NEXT(chrom, pos);
        NEXT(pos, id);
        NEXT(id, ref);
        NEXT(ref, alt);
        NEXT(alt, qual);
        NEXT(qual, filter);
        NEXT(filter, info);

        if (!chrom || !pos || !id || !ref || !alt || !qual || !filter)
        {
            log_error(stderr, "Error parsing VCF file (line %d): incomplete variant\n", line_counter);
            return false;
        }

#undef NEXT

        // convert position and quality
        char *endptr = NULL;
        uint32 position = strtoll(pos, &endptr, 10);
        if (!endptr || endptr == pos || *endptr != '\0')
        {
            log_error(stderr, "VCF file error (line %d): invalid position\n", line_counter);
            return false;
        }

        uint8 quality;
        if (*qual == '.')
        {
            quality = 0xff;
        } else {
            quality = (uint8) strtol(qual, &endptr, 10);
            if (!endptr || endptr == qual || *endptr != '\0')
            {
                log_warning(stderr, "VCF file error (line %d): invalid quality\n", line_counter);
                quality = 0xff;
            }
        }

        uint32 stop = position + strlen(ref);
        // parse the info header looking for a stop position
        if (info)
        {
            bool ret;
            ret = get_end_position(&stop, info);
            if (ret == false)
            {
                log_warning(stderr, "VCF file error (line %d): error parsing INFO line\n", line_counter);
                return false;
            }
        }

        // add an entry for each possible variant listed in this record
        do {
            char *next_base = strchr(alt, ',');
            if (next_base)
                *next_base = '\0';

            char *var;
            // if this is a called monomorphic variant (i.e., a site which has been identified as always having the same allele)
            // we store the reference string as the variant
            if (strcmp(alt, ".") == 0)
                var = ref;
            else
                var = alt;

            const uint32 ref_len = strlen(ref);
            const uint32 var_len = strlen(var);

            SNP_sequence_index index(output.reference_sequences.size(), ref_len,
                                     output.variants.size(), var_len);
            output.ref_variant_index.push_back(index);

            output.reference_sequence_names.push_back(std::string(chrom));
            output.sequence_positions.push_back(make_uint2(position, stop));

            output.reference_sequences.resize(index.reference_start + ref_len);
            bqsr::string_to_iupac16(ref, output.reference_sequences.begin() + index.reference_start);

            output.variants.resize(index.variant_start + var_len);
            bqsr::string_to_iupac16(var, output.variants.begin() + index.variant_start);

            output.variant_qualities.push_back(quality);

            if (next_base)
                alt = next_base + 1;
            else
                alt = NULL;
        } while (alt && *alt != '\0');
    }

    return true;
}
