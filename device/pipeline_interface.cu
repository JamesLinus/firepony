/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

#include "pipeline.h"

#include "alignment_data_device.h"
#include "sequence_data_device.h"
#include "variant_data_device.h"

#include "device/primitives/backends.h"

#include <thread>

#if ENABLE_TBB_BACKEND
#include <tbb/task_scheduler_init.h>
#endif

namespace firepony {

template <target_system system> void firepony_process_batch(firepony_context<system>& context, const alignment_batch<system>& batch);
template <target_system system> void firepony_postprocess(firepony_context<system>& context);

template <target_system system>
struct firepony_device_pipeline : public firepony_pipeline
{
    alignment_header<system> *header;
    sequence_data<system> *reference;
    variant_database<system> *dbsnp;

    firepony_context<system> *context;
    alignment_batch<system> *batch;

    io_thread *reader;

    std::thread thread;
    uint32 compute_device;

    firepony_device_pipeline(uint32 compute_device)
        : compute_device(compute_device)
    { }

    virtual std::string get_name(void) override;

    virtual target_system get_system(void) override
    {
        return system;
    }

    virtual int get_compute_device(void) override
    {
        return compute_device;
    }

    virtual pipeline_statistics& statistics(void) override
    {
        return context->stats;
    }

    virtual void setup(io_thread *reader,
                       const runtime_options *options,
                       alignment_header_host *h_header,
                       sequence_data_host *h_reference,
                       variant_database_host *h_dbsnp) override
    {
        size_t num_bytes;
        if (system == cuda)
        {
            hipSetDevice(compute_device);
        }

        this->reader = reader;

        header = new alignment_header<system>(*h_header);
        reference = new sequence_data<system>(*h_reference);
        dbsnp = new variant_database<system>(*h_dbsnp);

        header->download();

        num_bytes = reference->download();
        if (system == firepony::cuda)
        {
            fprintf(stderr, "downloaded %lu MB of reference data\n", num_bytes / (1024 * 1024));
        }

        num_bytes = dbsnp->download();
        if (system == firepony::cuda)
        {
            fprintf(stderr, "downloaded %lu MB of variant data\n", num_bytes / (1024 * 1024));
        }

        context = new firepony_context<system>(compute_device, *options, *header, *reference, *dbsnp);
        batch = new alignment_batch<system>();
    }

    virtual void start(void) override
    {
        thread = std::thread(&firepony_device_pipeline<system>::run, this);
    }

    virtual void join(void) override
    {
        thread.join();
    }

    {
    }

    virtual void postprocess(void) override
    {
        if (system == cuda)
        {
            hipSetDevice(compute_device);
        }

        firepony_postprocess(*context);
    }

private:
    void run(void)
    {
        if (system == cuda)
        {
            hipSetDevice(compute_device);
        }

        timer<host> io_timer;
        alignment_batch_host *h_batch;

        for(;;)
        {
            // try to get a batch to work on
            io_timer.start();
            h_batch = reader->get_batch();
            io_timer.stop();
            statistics().io.add(io_timer);

            if (h_batch == nullptr)
            {
                // no more data, we're done
                break;
            }

            // download to the device
            batch->download(h_batch);

            // process the batch
            firepony_process_batch(*context, *batch);

            // return it to the reader for reuse
            reader->retire_batch(h_batch);

            if (!context->options.debug)
            {
                fprintf(stderr, ".");
                fflush(stderr);
            }
        }
    }
};

#if ENABLE_CUDA_BACKEND
template<>
std::string firepony_device_pipeline<firepony::cuda>::get_name(void)
{
    hipDeviceProp_t prop;
    int runtime_version;

    hipRuntimeGetVersion(&runtime_version);
    hipSetDevice(compute_device);
    hipGetDeviceProperties(&prop, compute_device);

    char buf[1024];
    snprintf(buf, sizeof(buf),
             "%s (%lu MB, CUDA %d.%d)",
             prop.name, prop.totalGlobalMem / (1024 * 1024),
             runtime_version / 1000, runtime_version % 100);

    return std::string(buf);
}
#endif

#if ENABLE_CPP_BACKEND
template<>
std::string firepony_device_pipeline<firepony::cpp>::get_name(void)
{
    return std::string("CPU (C++ threads)");
}
#endif

#if ENABLE_OMP_BACKEND
template<>
std::string firepony_device_pipeline<firepony::omp>::get_name(void)
{
    return std::string("CPU (OpenMP)");
}
#endif

#if ENABLE_TBB_BACKEND
tbb::task_scheduler_init tbb_scheduler_init(tbb::task_scheduler_init::deferred);
static int num_tbb_threads = -1;

template<>
std::string firepony_device_pipeline<firepony::intel_tbb>::get_name(void)
{
    char buf[256];
    snprintf(buf, sizeof(buf), "CPU (Intel TBB, %d threads)", num_tbb_threads);
    return std::string(buf);
}

#endif

firepony_pipeline *firepony_pipeline::create(target_system system, uint32 device)
{
    switch(system)
    {
#if ENABLE_CUDA_BACKEND
    case firepony::cuda:
        return new firepony_device_pipeline<firepony::cuda>(device);
#endif

#if ENABLE_CPP_BACKEND
    case firepony::cpp:
        return new firepony_device_pipeline<firepony::cpp>();
#endif

#if ENABLE_OMP_BACKEND
    case firepony::omp:
        return new firepony_device_pipeline<firepony::omp>();
#endif

#if ENABLE_TBB_BACKEND
    case firepony::intel_tbb:
        // reserve device threads for other devices and I/O
        num_tbb_threads = tbb::task_scheduler_init::default_num_threads() - device - 1;
        tbb_scheduler_init.initialize(num_tbb_threads);
        return new firepony_device_pipeline<firepony::intel_tbb>(num_tbb_threads);
#endif

    default:
        return nullptr;
    }
}

} // namespace firepony
