#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

#include "../types.h"

#include "primitives/backends.h"
#include "primitives/hip/hip_runtime.h"
#include "primitives/parallel.h"
#include "primitives/util.h"

#include "util.h"

namespace firepony {

// the following two structures are the result of a couple of hours trying to do this with templates...
template <target_system system>
struct pack_uint8_to_2bit_vector
{
    typename d_packed_vector_2b<system>::view dest;
    typename d_vector_u8<system>::view src;

    pack_uint8_to_2bit_vector(typename d_packed_vector_2b<system>::view dest,
                              typename d_vector_u8<system>::view src)
        : dest(dest), src(src)
    { }

    CUDA_HOST_DEVICE void operator() (const uint32 word_index)
    {
        const uint8 *input = &src[word_index * d_packed_vector_2b<system>::SYMBOLS_PER_WORD];
        for(uint32 i = 0; i < d_packed_vector_2b<system>::SYMBOLS_PER_WORD; i++)
        {
            dest[word_index * d_packed_vector_2b<system>::SYMBOLS_PER_WORD + i] = input[i];
        }
    }
};

template <target_system system>
struct pack_uint8_to_1bit_vector
{
    typename d_packed_vector_1b<system>::view dest;
    typename d_vector_u8<system>::view src;

    pack_uint8_to_1bit_vector(typename d_packed_vector_1b<system>::view dest,
                              typename d_vector_u8<system>::view src)
        : dest(dest), src(src)
    { }

    CUDA_HOST_DEVICE void operator() (const uint32 word_index)
    {
        const uint8 *input = &src[word_index * d_packed_vector_1b<system>::SYMBOLS_PER_WORD];
        for(uint32 i = 0; i < d_packed_vector_1b<system>::SYMBOLS_PER_WORD; i++)
        {
            dest[word_index * d_packed_vector_1b<system>::SYMBOLS_PER_WORD + i] = input[i];
        }
    }
};

// prepare temp storage to store num_elements that will be packed into a bit vector
template <target_system system, typename packed_vector_dest>
static void pack_prepare_storage(d_vector_u8<system>& src, uint32 num_elements)
{
    src.resize(divide_ri(num_elements, packed_vector_dest::SYMBOLS_PER_WORD) * packed_vector_dest::SYMBOLS_PER_WORD);
}

// prepare temp_storage to store num_elements to be packed into a 1-bit vector
template <target_system system>
void pack_prepare_storage_2bit(d_vector_u8<system>& storage, uint32 num_elements)
{
    pack_prepare_storage<system, d_packed_vector_2b<system> >(storage, num_elements);
}
INSTANTIATE(pack_prepare_storage_2bit);

// prepare temp_storage to store num_elements to be packed into a 2-bit vector
template <target_system system>
void pack_prepare_storage_1bit(d_vector_u8<system>& storage, uint32 num_elements)
{
    pack_prepare_storage<system, d_packed_vector_1b<system> >(storage, num_elements);
}
INSTANTIATE(pack_prepare_storage_1bit);

template <target_system system>
void pack_to_2bit(d_packed_vector_2b<system>& dest, d_vector_u8<system>& src)
{
    dest.resize(src.size());
    parallel<system>::for_each(thrust::make_counting_iterator(0),
                               thrust::make_counting_iterator(0) + divide_ri(src.size(), d_packed_vector_2b<system>::SYMBOLS_PER_WORD),
                               pack_uint8_to_2bit_vector<system>(dest, src));
}
INSTANTIATE(pack_to_2bit);

template <target_system system>
void pack_to_1bit(d_packed_vector_1b<system>& dest, d_vector_u8<system>& src)
{
    dest.resize(src.size());
    parallel<system>::for_each(thrust::make_counting_iterator(0),
                               thrust::make_counting_iterator(0) + divide_ri(src.size(), d_packed_vector_1b<system>::SYMBOLS_PER_WORD),
                               pack_uint8_to_1bit_vector<system>(dest, src));
}
INSTANTIATE(pack_to_1bit);

// round a double to the Nth decimal place
// this is meant to workaround broken printf() rounding in glibc
double round_n(double val, int n)
{
    // xxxnsubtil: i suspect this might cause loss of precision if the initial exponent is large
    val = val * pow(10.0, n);
    val = round(val);
    val = val / pow(10.0, n);
    return val;
}

} // namespace firepony
