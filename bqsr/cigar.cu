#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

#include "from_nvbio/dna.h"

#include <thrust/iterator/transform_iterator.h>
#include <thrust/functional.h>

#include "cigar.h"
#include "bqsr_types.h"
#include "bqsr_context.h"
#include "alignment_data.h"
#include "util.h"

#include "primitives/hip/hip_runtime.h"
#include "primitives/parallel.h"

// compute the length of a given cigar operator
struct cigar_op_len : public thrust::unary_function<const cigar_op&, uint32>
{
    CUDA_HOST_DEVICE uint32 operator() (const cigar_op& op) const
    {
        return op.len;
    }
};

// expand cigar ops into temp storage
struct cigar_op_expand : public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE void operator() (const uint32 op_index)
    {
        const cigar_op& op = batch.cigars[op_index];
        const uint32 out_base = ctx.cigar.cigar_offsets[op_index];

        uint8 *out = &ctx.temp_storage[0] + out_base;

        for(uint32 i = 0; i < op.len; i++)
        {
            switch(op.op)
            {
            case cigar_op::OP_M:
            case cigar_op::OP_MATCH:
            case cigar_op::OP_X:
                out[i] = cigar_event::M;
                break;

            case cigar_op::OP_I:
            case cigar_op::OP_N:
                out[i] = cigar_event::I;
                break;

            case cigar_op::OP_D:
            case cigar_op::OP_H:
            case cigar_op::OP_P:
                out[i] = cigar_event::D;
                break;

            case cigar_op::OP_S:
                out[i] = cigar_event::S;
                break;
            }
        }
    }
};

// compact the cigar events from temporary storage into a 2-bit packed vector
struct cigar_op_compact : public bqsr_lambda
{
    cigar_op_compact(bqsr_context::view ctx,
                     const alignment_batch_device::const_view batch)
        : bqsr_lambda(ctx, batch)
    { }

    CUDA_HOST_DEVICE void operator() (const uint32 word_index)
    {
        D_PackedVector_2b::view& events = ctx.cigar.cigar_events;
        const uint8 *input = &ctx.temp_storage[word_index * D_PackedVector_2b::SYMBOLS_PER_WORD];

        for(uint32 i = 0; i < D_PackedVector_2b::SYMBOLS_PER_WORD; i++)
        {
            events[word_index * D_PackedVector_2b::SYMBOLS_PER_WORD + i] = input[i];
        }
    }
};

// expand cigar coordinates for a read
// xxxnsubtil: this is very similar to compute_alignment_window, should merge
struct cigar_coordinates_expand : public bqsr_lambda
{
    cigar_coordinates_expand(bqsr_context::view ctx,
                             const alignment_batch_device::const_view batch)
        : bqsr_lambda(ctx, batch)
    { }

    // update a coordinate window when we reach a new valid offset for the window
    template<typename W, typename O>
    CUDA_HOST_DEVICE void update(W& window, O new_offset,
                                 bool update_start = true,
                                 bool update_end = true)
    {
        if (update_start)
            window.x = bqsr::min(window.x, new_offset);

        if (update_end)
            window.y = bqsr::max(window.y, new_offset);
    }

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        const cigar_op *cigar = &batch.cigars[idx.cigar_start];

        uint32 base = ctx.cigar.cigar_offsets[idx.cigar_start];
        uint16 *output_read_coordinates = &ctx.cigar.cigar_event_read_coordinates[base];
        uint16 *output_reference_coordinates = &ctx.cigar.cigar_event_reference_coordinates[base];

        ushort2 read_window_clipped = make_ushort2(uint16(~0), 0);
        ushort2 read_window_clipped_no_insertions = make_ushort2(uint16(~0), 0);
        ushort2 reference_window_clipped = make_ushort2(uint16(~0), 0);

        uint16 read_offset = 0;
        uint16 reference_offset = 0;

        bool leading_clips = true;
        bool trailing_clips = false;

        for(uint32 c = 0; c < idx.cigar_len; c++)
        {
            switch(cigar[c].op)
            {
            case cigar_op::OP_M:
            case cigar_op::OP_MATCH:
            case cigar_op::OP_X:
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    leading_clips = false;

                    *output_read_coordinates++ = read_offset;
                    *output_reference_coordinates++ = reference_offset;

                    update(read_window_clipped, read_offset);
                    update(read_window_clipped_no_insertions, read_offset);
                    update(reference_window_clipped, reference_offset);

                    read_offset++;
                    reference_offset++;
                }

                break;

            case cigar_op::OP_S:
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    // if we're in a clipping region, then we're either in the leading or trailing clipping region
                    trailing_clips = !leading_clips;

                    *output_read_coordinates++ = read_offset;
                    *output_reference_coordinates++ = uint16(-1);

                    // if we haven't reached the trailing clipping region yet...
                    if (!trailing_clips)
                    {
                        // ... then update the end of the read windows
                        update(read_window_clipped, read_offset, false, true);
                        update(read_window_clipped_no_insertions, read_offset, false, true);
                    }

                    read_offset++;
                }

                break;

            case cigar_op::OP_N: // xxxnsubtil: N is really not supported and shouldn't be here
            case cigar_op::OP_I:
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    leading_clips = false;

                    *output_read_coordinates++ = read_offset;
                    *output_reference_coordinates++ = uint16(-1);

                    // update the trailing clipped read region
                    update(read_window_clipped, read_offset);

                    // the no-insertion window never moves with I

                    read_offset++;
                }

                break;

            case cigar_op::OP_D:
            case cigar_op::OP_H:
            case cigar_op::OP_P: // xxxnsubtil: not sure how to handle P
                for(uint32 i = 0; i < cigar[c].len; i++)
                {
                    *output_read_coordinates++ = uint16(-1);
                    *output_reference_coordinates++ = reference_offset;

                    update(reference_window_clipped, reference_offset);

                    reference_offset++;
                }
            }
        }

        ctx.cigar.read_window_clipped[read_index] = read_window_clipped;
        ctx.cigar.read_window_clipped_no_insertions[read_index] = read_window_clipped_no_insertions;
        ctx.cigar.reference_window_clipped[read_index] = reference_window_clipped;
    }
};

struct compute_error_vectors : public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        const bool negative_strand = batch.flags[read_index] & AlignmentFlags::REVERSE;

        // fetch the alignment base in reference coordinates
        const uint32 seq_id = batch.chromosome[read_index];
        const uint32 seq_base = ctx.reference.sequence_bp_start[seq_id];
        const uint32 align_offset = batch.alignment_start[read_index];
        const uint32 reference_alignment_start = seq_base + align_offset;

        uint16 current_bp_idx = 0;
        uint16 num_errors = 0;

        // go through the cigar events looking for the event we're interested in
        for(uint32 event = idx.cigar_start; event < idx.cigar_start + idx.cigar_len; event++)
        {
            // figure out the cigar event range for this event
            const uint32 cigar_start = ctx.cigar.cigar_offsets[event];
            const uint32 cigar_end = ctx.cigar.cigar_offsets[event+1];

            switch(batch.cigars[event].op)
            {
            case cigar_op::OP_M:

                for(uint32 i = cigar_start; i < cigar_end; i++)
                {
                    // update the current read bp index
                    current_bp_idx = ctx.cigar.cigar_event_read_coordinates[i];
                    // load the read bp
                    const uint8 read_bp = batch.reads[idx.read_start + current_bp_idx];

                    // load the corresponding sequence bp
                    const uint32 reference_bp_idx = reference_alignment_start + ctx.cigar.cigar_event_reference_coordinates[i];
                    const uint8 reference_bp = ctx.reference.bases[reference_bp_idx];

                    if (reference_bp != read_bp)
                    {
                        ctx.cigar.is_snp[idx.read_start + current_bp_idx] = 1;
                        num_errors++;
                    }
                }

                break;

            case cigar_event::I:
                // mark the read bp where an insertion begins
                current_bp_idx = ctx.cigar.cigar_event_read_coordinates[cigar_start];

                if (!negative_strand)
                {
                    ctx.cigar.is_insertion[idx.read_start + current_bp_idx - 1] = 1;
                } else {
                    ctx.cigar.is_insertion[idx.read_start + current_bp_idx + batch.cigars[event].len - 1] = 1;
                }

                num_errors++;
                break;

            case cigar_event::D:
                // note: deletions do not exist in the read, so current_bp_idx is not updated here
                // mark the read bp where a deletion begins
                if (!negative_strand)
                {
                    ctx.cigar.is_deletion[idx.read_start + current_bp_idx - 1] = 1;
                } else {
                    ctx.cigar.is_deletion[idx.read_start + current_bp_idx] = 1;
                }

                num_errors++;
                break;
            }
        }

        ctx.cigar.num_errors[read_index] = num_errors;
    }
};

#ifdef CUDA_DEBUG
// debug aid: sanity check that the expanded cigar events match what we expect
struct sanity_check_cigar_events : public bqsr_lambda
{
    sanity_check_cigar_events(bqsr_context::view ctx,
                              const alignment_batch_device::const_view batch)
        : bqsr_lambda(ctx, batch)
    { }

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        const cigar_op *cigar = &batch.cigars[idx.cigar_start];

        const uint32 cigar_start = ctx.cigar.cigar_offsets[idx.cigar_start];
        uint32 cigar_event_idx = 0;

        for(uint32 c = 0; c < idx.cigar_len; c++)
        {
            for(uint32 i = 0; i < cigar[c].len; i++)
            {
                switch(cigar[c].op)
                {
                case cigar_op::OP_M:
                case cigar_op::OP_MATCH:
                case cigar_op::OP_X:
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::M)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected M, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;

                case cigar_op::OP_N: // xxxnsubtil: N is really not supported and shouldn't be here
                case cigar_op::OP_I:
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::I)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected I, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;

                case cigar_op::OP_D:
                case cigar_op::OP_H:
                case cigar_op::OP_P: // xxxnsubtil: not sure how to handle P
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::D)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected D, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;

                case cigar_op::OP_S:
                    if (ctx.cigar.cigar_events[cigar_start + cigar_event_idx] != cigar_event::S)
                    {
                        printf("*** failed sanity check: read %u cigar op %u %u event offset %u: expected S, got %c\n",
                                read_index, c, i, cigar_start + cigar_event_idx, cigar_event::ascii(ctx.cigar.cigar_events[cigar_start + cigar_event_idx]));
                        return;
                    }

                    cigar_event_idx++;
                    break;
                }
            }
        }
    }
};
#endif

void expand_cigars(bqsr_context *context, const alignment_batch& batch)
{
    cigar_context& ctx = context->cigar;

    // compute the offsets of each expanded cigar op
    // xxxnsubtil: we ignore the active read list here, so we do unnecessary work
    // might want to revisit this
    ctx.cigar_offsets.resize(batch.device.cigars.size() + 1);

    // mark the first offset as 0
    thrust::fill_n(ctx.cigar_offsets.begin(), 1, 0);
    // do an inclusive scan to compute all offsets + the total size
    bqsr::inclusive_scan(thrust::make_transform_iterator(batch.device.cigars.begin(), cigar_op_len()),
                         batch.device.cigars.size(),
                         ctx.cigar_offsets.begin() + 1,
                         thrust::plus<uint32>());

    // read back the last element, which contains the size of the buffer required
    uint32 expanded_cigar_len = ctx.cigar_offsets[batch.device.cigars.size()];

    // make sure we have enough room for the expanded cigars
    // note: temporary storage must be padded to a multiple of the word size, since we'll pack whole words at a time
    pack_prepare_storage_2bit(context->temp_storage, expanded_cigar_len);
    ctx.cigar_events.resize(expanded_cigar_len);

    ctx.cigar_event_reference_coordinates.resize(expanded_cigar_len);
    ctx.cigar_event_read_coordinates.resize(expanded_cigar_len);

    ctx.read_window_clipped.resize(batch.device.num_reads);
    ctx.read_window_clipped_no_insertions.resize(batch.device.num_reads);
    ctx.reference_window_clipped.resize(batch.device.num_reads);

    ctx.is_snp.resize(batch.device.reads.size());
    ctx.is_insertion.resize(batch.device.reads.size());
    ctx.is_deletion.resize(batch.device.reads.size());
    ctx.num_errors.resize(batch.device.num_reads);

    // the following require zero initialization
    thrust::fill(ctx.is_snp.m_storage.begin(), ctx.is_snp.m_storage.end(), 0);
    thrust::fill(ctx.is_insertion.m_storage.begin(), ctx.is_insertion.m_storage.end(), 0);
    thrust::fill(ctx.is_deletion.m_storage.begin(), ctx.is_deletion.m_storage.end(), 0);
    thrust::fill(ctx.num_errors.begin(), ctx.num_errors.end(), 0);

    // expand the cigar ops into temp storage (xxxnsubtil: same as above, active read list is ignored)
    thrust::for_each(thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(0) + batch.device.cigars.size(),
                     cigar_op_expand(*context, batch.device));

    // pack the cigar into a 2-bit vector
    pack_to_2bit(ctx.cigar_events, context->temp_storage);

#ifdef CUDA_DEBUG
    thrust::for_each(context->active_read_list.begin(),
                     context->active_read_list.end(),
                     sanity_check_cigar_events(*context, batch.device));
#endif

    // now expand the coordinates per read
    // this avoids having to deal with boundary conditions within reads
    thrust::for_each(context->active_read_list.begin(),
                     context->active_read_list.end(),
                     cigar_coordinates_expand(*context, batch.device));

    // compute the error bit vectors
    // this also counts the number of errors in each read
    thrust::for_each(context->active_read_list.begin(),
                     context->active_read_list.end(),
                     compute_error_vectors(*context, batch.device));
}

void debug_cigar(bqsr_context *context, const alignment_batch& batch, int read_index)
{
    const alignment_batch_host& h_batch = batch.host;

    const CRQ_index idx = h_batch.crq_index(read_index);
    const cigar_context& ctx = context->cigar;

    printf("  cigar info:\n");

    printf("    cigar                       = [");
    for(uint32 i = idx.cigar_start; i < idx.cigar_start + idx.cigar_len; i++)
    {
        cigar_op op = h_batch.cigars[i];
        printf("%d%c", op.len, op.ascii_op());
    }
    printf("]\n");

    uint32 cigar_start = ctx.cigar_offsets[idx.cigar_start];
    uint32 cigar_end = ctx.cigar_offsets[idx.cigar_start + idx.cigar_len];
    printf("    offset range                = [% 3d, % 3d]\n", cigar_start, cigar_end);

    printf("                                    ");
    for(uint32 i = 0; i < cigar_end - cigar_start; i++)
    {
        printf("% 3d ", i);
    }
    printf("\n");

    printf("    event list                  = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        printf("  %c ", cigar_event::ascii(ctx.cigar_events[i]));
    }
    printf("]\n");

    printf("    event idx -> read coords    = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        printf("% 3d ", (int16) ctx.cigar_event_read_coordinates[i]);
    }
    printf("]\n");

    printf("    event reference coordinates = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        printf("% 3d ", (int16) ctx.cigar_event_reference_coordinates[i]);
    }
    printf("]\n");

    printf("    is snp                      = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 read_bp_idx = ctx.cigar_event_read_coordinates[i];
        if (read_bp_idx == uint16(-1))
        {
            printf("  - ");
        } else {
            printf("% 3d ", (uint8) ctx.is_snp[idx.read_start + read_bp_idx]);
        }
    }
    printf("]\n");

    printf("    is insertion                = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 read_bp_idx = ctx.cigar_event_read_coordinates[i];
        if (read_bp_idx == uint16(-1))
        {
            printf("  - ");
        } else {
            printf("% 3d ", (uint8) ctx.is_insertion[read_bp_idx]);
        }
    }
    printf("]\n");

    printf("    is deletion                 = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 read_bp_idx = ctx.cigar_event_read_coordinates[i];
        if (read_bp_idx == uint16(-1))
        {
            printf("  - ");
        } else {
            printf("% 3d ", (uint8) ctx.is_deletion[read_bp_idx]);
        }
    }
    printf("]\n");

    printf("    active location list        = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        uint16 bp_offset = ctx.cigar_event_read_coordinates[i];
        if (bp_offset == uint16(-1))
        {
            printf("  - ");
        } else {
            printf("% 3d ", context->active_location_list[idx.read_start + bp_offset] ? 1 : 0);
        }
    }
    printf("]\n");

    const uint32 ref_sequence_id = h_batch.chromosome[read_index];
    const uint32 ref_sequence_base = context->reference.host.sequence_bp_start[ref_sequence_id];
    const uint32 ref_sequence_offset = ref_sequence_base + h_batch.alignment_start[read_index];

    printf("    reference sequence data     = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 ref_bp = ctx.cigar_event_reference_coordinates[i];
        printf("  %c ", ref_bp == uint16(-1) ? '-' : from_nvbio::iupac16_to_char(context->reference.host.bases[ref_sequence_offset + ref_bp]));
    }
    printf("]\n");

    printf("    read sequence data          = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 read_bp = ctx.cigar_event_read_coordinates[i];
        char base;

        if (read_bp == uint16(-1))
        {
            base = '-';
        } else {
            base = from_nvbio::iupac16_to_char(h_batch.reads[idx.read_start + read_bp]);
            if (ctx.cigar_events[i] == cigar_event::S)
            {
                // display soft-clipped bases in lowercase
                base = tolower(base);
            }
        }

        printf("  %c ", base);
    }
    printf("]\n");

    printf("    read quality data           = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 read_bp = ctx.cigar_event_read_coordinates[i];

        if (read_bp == uint16(-1))
        {
            printf("    ");
        } else {
            printf("% 3d ", h_batch.qualities[idx.qual_start + read_bp]);
        }
    }
    printf("]\n");

    printf("    ... in ascii                = [ ");
    for(uint32 i = cigar_start; i < cigar_end; i++)
    {
        const uint16 read_bp = ctx.cigar_event_read_coordinates[i];

        if (read_bp == uint16(-1))
        {
            printf(" -  ");
        } else {
            printf("  %c ", h_batch.qualities[idx.qual_start + read_bp] + '!');
        }
    }
    printf("]\n");

    ushort2 read_window_clipped = ctx.read_window_clipped[read_index];
    printf("    clipped read window         = [ % 3d, % 3d ]\n", read_window_clipped.x, read_window_clipped.y);

    ushort2 read_window_clipped_no_insertions = ctx.read_window_clipped_no_insertions[read_index];
    printf("    ... lead/trail insertions   = [ % 3d, % 3d ]\n",
                read_window_clipped_no_insertions.x, read_window_clipped_no_insertions.y);

    ushort2 reference_window_clipped = ctx.reference_window_clipped[read_index];
    printf("    clipped reference window    = [ % 3d, % 3d ]\n",
                reference_window_clipped.x, reference_window_clipped.y);

    uint16 err = ctx.num_errors[read_index];
    printf("    number of errors            = [ % 3d ]\n", err);

    printf("\n");
}
