#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

// base alignment quality calculations (gatk: BAQ.java)

#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/functional.h>

#include <stdlib.h>
#include <math.h>

#include "bqsr_types.h"
#include "alignment_data.h"
#include "sequence_data.h"
#include "bqsr_context.h"

#include "primitives/util.h"
#include "primitives/parallel.h"
#include "from_nvbio/dna.h"
#include "from_nvbio/alphabet.h"

#define MAX_PHRED_SCORE 93
#define EM 0.33333333333
#define EI 0.25

#define MAX_BAND_WIDTH 7
#define MIN_BASE_QUAL 4

// all bases with q < minBaseQual are up'd to this value
#define MIN_BASE_QUAL 4

#define GAP_OPEN_PROBABILITY (pow(10.0, (-40.0)/10.))
#define GAP_EXTENSION_PROBABILITY 0.1

struct compute_hmm_windows : public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        uint2&   out_reference_window = ctx.baq.reference_windows[read_index];

        // grab reference sequence window in the genome
        const uint32 ref_ID = batch.chromosome[read_index];
        const uint32 ref_base = ctx.reference.sequence_bp_start[ref_ID];
        const uint32 ref_length = ctx.reference.sequence_bp_len[ref_ID];

        const uint32 seq_to_alignment_offset = batch.alignment_start[read_index];

        const ushort2& read_window = ctx.cigar.read_window_clipped[read_index];
        const ushort2& read_window_no_insertions = ctx.cigar.read_window_clipped_no_insertions[read_index];
        const ushort2& reference_window = ctx.cigar.reference_window_clipped[read_index];

        const uint32 first_insertion_offset = read_window_no_insertions.x - read_window.x;
        const uint32 last_insertion_offset = read_window_no_insertions.y - read_window.y;

        const int offset = MAX_BAND_WIDTH / 2;
        uint32 readStart = reference_window.x + seq_to_alignment_offset; // always clipped

        // reference window for HMM
        uint32 start = bqsr::max(readStart - offset - first_insertion_offset, 0u);
        uint32 stop = reference_window.y + seq_to_alignment_offset + offset + last_insertion_offset;

        if (stop > ref_length)
        {
            out_reference_window = make_uint2(uint32(-1), uint32(-1));
            return;
        }

        start += ref_base;
        stop += ref_base;

        out_reference_window = make_uint2(start, stop);
    }
};

// encapsulates common state for the HMM algorithm
struct hmm_common : public bqsr_lambda
{
    D_VectorU32::view baq_state;

    hmm_common(bqsr_context::view ctx,
               const alignment_batch_device::const_view batch,
               D_VectorU32::view baq_state)
        : bqsr_lambda(ctx, batch), baq_state(baq_state)
    { }

    int bandWidth, bandWidth2;

    int referenceStart, referenceLength;
    int queryStart, queryEnd, queryLen;

    double *forwardMatrix;
    double *backwardMatrix;
    double *scalingFactors;

    double sM, sI, bM, bI;

    double m[9];

    D_StreamDNA16 referenceBases;
    D_StreamDNA16 queryBases;
    const uint8 *inputQualities;

    uint8 *outputQualities;
    uint32 *outputState;

    template<typename Tuple>
    CUDA_HOST_DEVICE void setup(const Tuple& hmm_index)
    {
        const uint32 read_index    = thrust::get<0>(hmm_index);
        const uint32 matrix_index  = thrust::get<1>(hmm_index);
        const uint32 scaling_index = thrust::get<2>(hmm_index);

        const CRQ_index idx = batch.crq_index(read_index);

        // set up matrix and scaling factor pointers
        forwardMatrix = &ctx.baq.forward[matrix_index];
        backwardMatrix = &ctx.baq.backward[matrix_index];
        scalingFactors = &ctx.baq.scaling[scaling_index];

        // get the windows for the current read
        const uint2& reference_window = ctx.baq.reference_windows[read_index];
        const ushort2& read_window = ctx.cigar.read_window_clipped[read_index];

        referenceStart = reference_window.x;
        referenceLength = reference_window.y - reference_window.x + 1;

        queryStart = read_window.x;
        queryEnd = read_window.y;
        queryLen = read_window.y - read_window.x + 1;

        // compute band width
        if (referenceLength > queryLen)
            bandWidth = referenceLength;
        else
            bandWidth = queryLen;

        if (MAX_BAND_WIDTH < abs(referenceLength - queryLen))
        {
            bandWidth = abs(referenceLength - queryLen) + 3;
        }

        if (bandWidth > MAX_BAND_WIDTH)
            bandWidth = MAX_BAND_WIDTH;

        if (bandWidth < abs(referenceLength - queryLen))
        {
            bandWidth = abs(referenceLength - queryLen);
        }

        bandWidth2 = bandWidth * 2 + 1;

        // initialize transition probabilities
        sM = 1.0 / (2 * queryLen + 2);
        sI = sM;
        bM = (1 - GAP_OPEN_PROBABILITY) / referenceLength;
        bI = GAP_OPEN_PROBABILITY / referenceLength;

        m[0*3+0] = (1 - GAP_OPEN_PROBABILITY - GAP_OPEN_PROBABILITY) * (1 - sM);
        m[0*3+1] = GAP_OPEN_PROBABILITY * (1 - sM);
        m[0*3+2] = m[0*3+1];
        m[1*3+0] = (1 - GAP_EXTENSION_PROBABILITY) * (1 - sI);
        m[1*3+1] = GAP_EXTENSION_PROBABILITY * (1 - sI);
        m[1*3+2] = 0.0;
        m[2*3+0] = 1 - GAP_EXTENSION_PROBABILITY;
        m[2*3+1] = 0.0;
        m[2*3+2] = GAP_EXTENSION_PROBABILITY;

//        printf("referenceStart = %u\n", referenceStart);
//        printf("queryStart = %u queryLen = %u\n", queryStart, queryLen);

        queryBases = batch.reads + idx.read_start + queryStart;
        referenceBases = ctx.reference.bases + referenceStart;
        inputQualities = &batch.qualities[idx.qual_start] + queryStart;

        if (ctx.baq.qualities.size() > 0)
            outputQualities = &ctx.baq.qualities[idx.qual_start] + queryStart;
        else
            outputQualities = NULL;

        if (baq_state.size() > 0)
            outputState = &baq_state[idx.qual_start] + queryStart;
        else
            outputState = NULL;

        queryStart = 0;
    }

    CUDA_HOST_DEVICE int set_u(const int b, const int i, const int k)
    {
        int x = i - b;
        x = x > 0 ? x : 0;
        return (k + 1 - x) * 3;
    }

    // computes a matrix offset for forwardMatrix or backwardMatrix
    CUDA_HOST_DEVICE int off(int i, int j = 0)
    {
        return i * 6 * (2 * MAX_BAND_WIDTH + 1) + j;
    }

    // computes the required HMM matrix size for the given read length
    CUDA_HOST_DEVICE static uint32 matrix_size(const uint32 read_len)
    {
        return (read_len + 1) * 6 * (2 * MAX_BAND_WIDTH + 1);
    }

    CUDA_HOST_DEVICE static double qual2prob(uint8 q)
    {
        return pow(10.0, -q/10.0);
    }

    CUDA_HOST_DEVICE static double calcEpsilon(uint8 ref, uint8 read, uint8 qualB)
    {
        if (ref == from_nvbio::AlphabetTraits<from_nvbio::DNA_IUPAC>::N ||
            read == from_nvbio::AlphabetTraits<from_nvbio::DNA_IUPAC>::N)
        {
            return 1.0;
        }

        double qual = qual2prob(qualB < MIN_BASE_QUAL ? MIN_BASE_QUAL : qualB);
        double e = (ref == read ? 1 - qual : qual * EM);
        return e;
    }
};

struct hmm_glocal_forward : public hmm_common
{
    using hmm_common::hmm_common;

    template<typename Tuple>
    CUDA_HOST_DEVICE void operator() (const Tuple& hmm_index)
    {
        int i, k;

        hmm_common::setup(hmm_index);

//        const uint32 read_index    = thrust::get<0>(hmm_index);
//        printf("read %d: hmm_glocal(l_ref=%d qstart=%d, l_query=%d)\n", read_index, referenceLength, queryStart, queryLen);
//        printf("read %d: ref = { ", read_index);
//        for(int c = 0; c < referenceLength; c++)
//        {
//            printf("%c ", from_nvbio::iupac16_to_char(referenceBases[c]));
//        }
//        printf("\n");
//
//        printf("read %d: que = { ", read_index);
//        for(int c = 0; c < queryLen; c++)
//        {
//            printf("%c ", from_nvbio::iupac16_to_char(queryBases[c]));
//        }
//        printf("\n");

//        printf("read %d: _iqual = { % 3d % 3d % 3d % 3d % 3d ... % 3d % 3d % 3d % 3d % 3d }\n", read_index,
//                inputQualities[0], inputQualities[1], inputQualities[2], inputQualities[3], inputQualities[4],
//                inputQualities[queryLen - 5], inputQualities[queryLen - 4], inputQualities[queryLen - 3], inputQualities[queryLen - 2], inputQualities[queryLen - 1]);
//        printf("read %d: c->bw = %d, bw = %d, l_ref = %d, l_query = %d\n", read_index, MAX_BAND_WIDTH, bandWidth, referenceLength, queryLen);

        /*** forward ***/
        // f[0]
        forwardMatrix[off(0, set_u(bandWidth, 0, 0))] = 1.0;
        scalingFactors[0] = 1.0;
        { // f[1]
            double *fi = &forwardMatrix[off(1)];
            double sum;
            int beg = 1;
            int end = referenceLength < bandWidth + 1? referenceLength : bandWidth + 1;
            int _beg, _end;

            sum = 0.0;
            for (k = beg; k <= end; ++k)
            {
                int u;
                double e = calcEpsilon(referenceBases[k-1], queryBases[queryStart], inputQualities[queryStart]);
//                printf("read %d: referenceBases[%d-1] = %c inputQualities[%d] = %d queryBases[%d] = %c -> e = %.4f\n",
//                        read_index,
//                        k,
//                        from_nvbio::iupac16_to_char(referenceBases[k-1]),
//                        queryStart,
//                        inputQualities[queryStart],
//                        queryStart,
//                        from_nvbio::iupac16_to_char(queryBases[queryStart]), e);

                u = set_u(bandWidth, 1, k);

                fi[u+0] = e * bM;
                fi[u+1] = EI * bI;

                sum += fi[u] + fi[u+1];
            }

            // rescale
            scalingFactors[1] = sum;
            _beg = set_u(bandWidth, 1, beg);
            _end = set_u(bandWidth, 1, end);
            _end += 2;

            for (int k = _beg; k <= _end; ++k)
                fi[k] /= sum;
        }

        // f[2..l_query]
        for (i = 2; i <= queryLen; ++i)
        {
            double *fi = &forwardMatrix[off(i)];
            double *fi1 = &forwardMatrix[off(i-1)];
            double sum;

            int beg = 1;
            int end = referenceLength;
            int x, _beg, _end;

            char qyi = queryBases[queryStart+i-1];

            x = i - bandWidth;
            beg = beg > x? beg : x; // band start

            x = i + bandWidth;
            end = end < x? end : x; // band end

            sum = 0.0;
            for (k = beg; k <= end; ++k)
            {
                int u, v11, v01, v10;
                double e = calcEpsilon(referenceBases[k-1], qyi, inputQualities[queryStart+i-1]);
//                printf("read %d: referenceBases[%d-1] = %c inputQualities[%d+%d-1] = %d qyi = %c -> e = %.4f\n",
//                        read_index,
//                        k,
//                        from_nvbio::iupac16_to_char(referenceBases[k-1]),
//                        queryStart,
//                        i,
//                        inputQualities[queryStart+i-1],
//                        from_nvbio::iupac16_to_char(qyi), e);

                u = set_u(bandWidth, i, k);
                v11 = set_u(bandWidth, i-1, k-1);
                v10 = set_u(bandWidth, i-1, k);
                v01 = set_u(bandWidth, i, k-1);

                fi[u+0] = e * (m[0] * fi1[v11+0] + m[3] * fi1[v11+1] + m[6] * fi1[v11+2]);
                fi[u+1] = EI * (m[1] * fi1[v10+0] + m[4] * fi1[v10+1]);
                fi[u+2] = m[2] * fi[v01+0] + m[8] * fi[v01+2];

                sum += fi[u] + fi[u+1] + fi[u+2];

    //            printf("(%d,%d;%d): %.4f,%.4f,%.4f\n", i, k, u, fi[u], fi[u+1], fi[u+2]);
    //            printf(" .. u = %d v11 = %d v01 = %d v10 = %d e = %f\n", u, v11, v01, v10, e);
            }

            // rescale
            scalingFactors[i] = sum;

            _beg = set_u(bandWidth, i, beg);
            _end = set_u(bandWidth, i, end);
            _end += 2;

            for (k = _beg, sum = 1./sum; k <= _end; ++k)
                fi[k] *= sum;
        }

        { // f[l_query+1]
            double sum = 0.0;

            for (k = 1; k <= referenceLength; ++k)
            {
                int u = set_u(bandWidth, queryLen, k);

                if (u < 3 || u >= bandWidth2*3+3)
                    continue;

                sum += forwardMatrix[off(queryLen,u+0)] * sM + forwardMatrix[off(queryLen, u+1)] * sI;
            }

            scalingFactors[queryLen+1] = sum; // the last scaling factor
        }
    }
};

struct hmm_glocal_backward : public hmm_common
{
    using hmm_common::hmm_common;

    template<typename Tuple>
    CUDA_HOST_DEVICE void operator() (const Tuple& hmm_index)
    {
        int i, k;

        hmm_common::setup(hmm_index);

        /*** backward ***/
        // b[l_query] (b[l_query+1][0]=1 and thus \tilde{b}[][]=1/s[l_query+1]; this is where s[l_query+1] comes from)
        for (k = 1; k <= referenceLength; ++k)
        {
            int u = set_u(bandWidth, queryLen, k);
            double *bi = &backwardMatrix[off(queryLen)];

            if (u < 3 || u >= bandWidth2*3+3)
                continue;

            bi[u+0] = sM / scalingFactors[queryLen] / scalingFactors[queryLen+1];
            bi[u+1] = sI / scalingFactors[queryLen] / scalingFactors[queryLen+1];
        }

        // b[l_query-1..1]
        for (i = queryLen - 1; i >= 1; --i)
        {
            int beg = 1;
            int end = referenceLength;
            int x, _beg, _end;

            double *bi = &backwardMatrix[off(i)];
            double *bi1 = &backwardMatrix[off(i+1)];
            double y = (i > 1)? 1. : 0.;

            char qyi1 = queryBases[queryStart+i];

            x = i - bandWidth;
            beg = beg > x? beg : x;

            x = i + bandWidth;
            end = end < x? end : x;

            for (k = end; k >= beg; --k)
            {
                int u, v11, v01, v10;

                u = set_u(bandWidth, i, k);
                v11 = set_u(bandWidth, i+1, k+1);
                v10 = set_u(bandWidth, i+1, k);
                v01 = set_u(bandWidth, i, k+1);

                /* const */ double e;
                if (k >= referenceLength)
                    e = 0;
                else
                    e = calcEpsilon(referenceBases[k], qyi1, inputQualities[queryStart+i]) * bi1[v11];

                bi[u+0] = e * m[0] + EI * m[1] * bi1[v10+1] + m[2] * bi[v01+2]; // bi1[v11] has been folded into e.
                bi[u+1] = e * m[3] + EI * m[4] * bi1[v10+1];
                bi[u+2] = (e * m[6] + m[8] * bi[v01+2]) * y;
            }

            // rescale
            _beg = set_u(bandWidth, i, beg);
            _end = set_u(bandWidth, i, end);
            _end += 2;

            y = 1.0 / scalingFactors[i];
            for (k = _beg; k <= _end; ++k)
                bi[k] *= y;
        }

//        double pb = 0.0;
        { // b[0]
            int beg = 1;
            int end = referenceLength < bandWidth + 1? referenceLength : bandWidth + 1;

            double sum = 0.0;
            for (k = end; k >= beg; --k)
            {
                int u = set_u(bandWidth, 1, k);
                double e = calcEpsilon(referenceBases[k-1], queryBases[queryStart], inputQualities[queryStart]);

                if (u < 3 || u >= bandWidth2*3+3)
                    continue;

                sum += e * backwardMatrix[off(1, u+0)] * bM + EI * backwardMatrix[off(1, u+1)] * bI;
            }

            backwardMatrix[off(0, set_u(bandWidth, 0, 0))] = sum / scalingFactors[0];
//            pb = backwardMatrix[off(0, set_u(bandWidth, 0, 0))]; // if everything works as is expected, pb == 1.0
        }
    }
};

struct hmm_glocal_map : public hmm_common
{
    using hmm_common::hmm_common;

    template<typename Tuple>
    CUDA_HOST_DEVICE void operator() (const Tuple& hmm_index)
    {
        int i, k;

        hmm_common::setup(hmm_index);

        /*** MAP ***/
        for (i = 1; i <= queryLen; ++i)
        {
            double sum = 0.0;
            double max = 0.0;

            const double *fi = &forwardMatrix[off(i)];
            const double *bi = &backwardMatrix[off(i)];

            int beg = 1;
            int end = referenceLength;
            int x, max_k = -1;

            x = i - bandWidth;
            beg = beg > x? beg : x;

            x = i + bandWidth;
            end = end < x? end : x;

            for (k = beg; k <= end; ++k)
            {
                const int u = set_u(bandWidth, i, k);
                double z = 0.0;

                z = fi[u+0] * bi[u+0];
                sum += z;
                if (z > max)
                {
                    max = z;
                    max_k = (k-1) << 2 | 0;
                }

                z = fi[u+1] * bi[u+1];
                sum += z;
                if (z > max)
                {
                    max = z;
                    max_k = (k-1) << 2 | 1;
                }
            }

            max /= sum;
            sum *= scalingFactors[i]; // if everything works as is expected, sum == 1.0

            if (outputState != NULL)
                outputState[queryStart+i-1] = max_k;

            if (outputQualities != NULL)
            {
                k = (int)(double(-4.343) * log(double(1.0) - double(max)) + double(.499)); // = 10*log10(1-max)
                outputQualities[queryStart+i-1] = (char)(k > 100? 99 : (k < MIN_BASE_QUAL ? MIN_BASE_QUAL : k));

//                printf("read %d: outputQualities[%d]: max = %.16f k = %d -> %d\n", read_index, queryStart+i-1, max, k, outputQualities[queryStart+i-1]);
            }

    //        printf("(%.4f,%.4f) (%d,%d,%d,%.4f)\n", pb, sum, (i-1), (max_k>>2), (max_k&3), max);
        }
    }
};

// functor to compute the size required for the forward/backward HMM matrix
// note that this computes the size required for *one* matrix only; we allocate the matrices on two separate vectors and use the same index for both
struct compute_hmm_matrix_size : public thrust::unary_function<uint32, uint32>, public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE uint32 operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        return hmm_common::matrix_size(idx.read_len);
    }
};

struct compute_hmm_scaling_factor_size : public thrust::unary_function<uint32, uint32>, public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE uint32 operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);
        return idx.read_len + 2;
    }
};

struct read_needs_baq : public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE bool operator() (const uint32 read_index)
    {
        if (ctx.cigar.num_errors[read_index] != 0)
            return true;

        return false;
    }
};

struct read_flat_baq : public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        if (ctx.baq.qualities.size() == 0)
        {
            return;
        }

        if (ctx.cigar.num_errors[read_index] != 0)
        {
            // reads with errors will have BAQ computed explicitly
            return;
        }

        const CRQ_index idx = batch.crq_index(read_index);
        const ushort2& read_window = ctx.cigar.read_window_clipped[read_index];
        const uint32 queryStart = read_window.x;
        const uint32 queryLen = read_window.y - read_window.x + 1;
        uint8 *outputQualities = &ctx.baq.qualities[idx.qual_start] + queryStart;

        memset(outputQualities, NO_BAQ_UNCERTAINTY, queryLen);
    }
};

// bottom half of BAQ.calcBAQFromHMM in GATK
struct cap_baq_qualities : public bqsr_lambda
{
    D_VectorU32::view baq_state;

    cap_baq_qualities(bqsr_context::view ctx,
                      const alignment_batch_device::const_view batch,
                      D_VectorU32::view baq_state)
        : bqsr_lambda(ctx, batch), baq_state(baq_state)
    { }

    CUDA_HOST_DEVICE bool stateIsIndel(uint32 state)
    {
        return (state & 3) != 0;
    }

    CUDA_HOST_DEVICE uint32 stateAlignedPosition(uint32 state)
    {
        return state >> 2;
    }

    CUDA_HOST_DEVICE uint8 capBaseByBAQ(uint8 oq, uint8 bq, uint32 state, uint32 expectedPos)
    {
        uint8 b;
        bool isIndel = stateIsIndel(state);
        uint32 pos = stateAlignedPosition(state);

        if (isIndel || pos != expectedPos) // we are an indel or we don't algin to our best current position
        {
            b = MIN_BASE_QUAL; // just take b = minBaseQuality
        } else {
            b = bqsr::min(bq, oq);
        }

        return b;
    }

    // xxxnsubtil: this could use some cleanup
    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);

        const uint32 cigar_start = ctx.cigar.cigar_offsets[idx.cigar_start];
        const uint32 cigar_end = ctx.cigar.cigar_offsets[idx.cigar_start + idx.cigar_len];

        const ushort2& read_window = ctx.cigar.read_window_clipped[read_index];
        const ushort2& read_window_no_insertions = ctx.cigar.read_window_clipped_no_insertions[read_index];
        const ushort2& reference_window = ctx.cigar.reference_window_clipped[read_index];

        const uint32 seq_to_alignment_offset = batch.alignment_start[read_index];
        const uint32 first_insertion_offset = read_window_no_insertions.x - read_window.x;

        const int offset = MAX_BAND_WIDTH / 2;

        const uint32 readStart = reference_window.x + seq_to_alignment_offset;
        const uint32 start = bqsr::max(readStart - offset - first_insertion_offset, 0u);

        const int refOffset = (int)(start - readStart);

        uint32 readI = 0;
        uint32 refI = 0;
        uint32 current_op_offset = 0;

        // scan for the start of the baq region
        uint32 i;
        for(i = 0; i < cigar_end - cigar_start; i++)
        {
            const uint16 read_bp_idx = ctx.cigar.cigar_event_read_coordinates[cigar_start + i];
            if (read_bp_idx >= read_window.x)
                break;
        }

        const uint32 baq_start = i;

        for(; i < cigar_end - cigar_start; i++)
        {
            const uint16 read_bp_idx = ctx.cigar.cigar_event_read_coordinates[cigar_start + i];
            const uint32 qual_idx = idx.qual_start + read_bp_idx;

            switch(ctx.cigar.cigar_events[i + cigar_start])
            {
            case cigar_event::S:
                refI++;
                current_op_offset = 0;
                break;

            case cigar_event::I:
                ctx.baq.qualities[qual_idx] = batch.qualities[qual_idx];
                readI++;
                current_op_offset = 0;
                break;

            case cigar_event::D:
                refI++;
                current_op_offset = 0;
                break;

            case cigar_event::M:
                const uint32 expectedPos = refI - refOffset + (i - baq_start - readI);
                ctx.baq.qualities[qual_idx] = capBaseByBAQ(batch.qualities[idx.qual_start + read_bp_idx],
                                                           ctx.baq.qualities[idx.qual_start + read_bp_idx],
                                                           baq_state[idx.qual_start + read_bp_idx],
                                                           expectedPos);
                readI++;
                refI++;
                current_op_offset++;

                break;
            }
        }
    }
};

// transforms BAQ scores the same way as GATK's encodeBQTag
struct recode_baq_qualities : public bqsr_lambda
{
    using bqsr_lambda::bqsr_lambda;

    CUDA_HOST_DEVICE void operator() (const uint32 read_index)
    {
        const CRQ_index idx = batch.crq_index(read_index);

        for(uint32 i = idx.qual_start; i < idx.qual_start + idx.qual_len; i++)
        {
            const uint8 baq_i = ctx.baq.qualities[i];
            if (baq_i == uint8(-1))
            {
                continue;
            }

            const uint8 bq = batch.qualities[i] + 64;
            const uint8 tag = bq - baq_i;
            ctx.baq.qualities[i] = tag;
        }
    }
};

void baq_reads(bqsr_context *context, const alignment_batch& batch)
{
    struct baq_context& baq = context->baq;
    D_VectorU32& active_baq_read_list = context->temp_u32;
    D_VectorU32& baq_state = context->temp_u32_2;

    uint32 num_active;

    // collect the reads that we need to compute BAQ for
    active_baq_read_list.resize(context->active_read_list.size());

    num_active = bqsr::copy_if(context->active_read_list.begin(),
                               context->active_read_list.size(),
                               active_baq_read_list.begin(),
                               read_needs_baq(*context, batch.device),
                               context->temp_storage);

    active_baq_read_list.resize(num_active);

    // compute the index and size of the HMM matrices
    baq.matrix_index.resize(num_active + 1);
    // first offset is zero
    thrust::fill_n(baq.matrix_index.begin(), 1, 0);
    // do an inclusive scan to compute all offsets + the total size
    bqsr::inclusive_scan(thrust::make_transform_iterator(active_baq_read_list.begin(),
                                                         compute_hmm_matrix_size(*context, batch.device)),
                         num_active,
                         baq.matrix_index.begin() + 1,
                         thrust::plus<uint32>());

    // compute the index and size of the HMM scaling factors
    baq.scaling_index.resize(num_active + 1);
    // first offset is zero
    thrust::fill_n(baq.scaling_index.begin(), 1, 0);
    bqsr::inclusive_scan(thrust::make_transform_iterator(active_baq_read_list.begin(),
                                                         compute_hmm_scaling_factor_size(*context, batch.device)),
                         num_active,
                         baq.scaling_index.begin() + 1,
                         thrust::plus<uint32>());

    // read back the last elements, which contain the size of the buffer required
    uint32 matrix_len = baq.matrix_index[num_active];
    uint32 scaling_len = baq.scaling_index[num_active];

//    printf("reads: %u\n", batch.num_reads);
//    printf("forward len = %u bytes = %lu\n", matrix_len, matrix_len * sizeof(double));
//    printf("expected len = %lu expected bytes = %lu\n",
//            hmm_common::matrix_size(100) * context->active_read_list.size(),
//            hmm_common::matrix_size(100) * context->active_read_list.size() * sizeof(double));
//    printf("per read matrix size = %u bytes = %lu\n", hmm_common::matrix_size(100), hmm_common::matrix_size(100) * sizeof(double));

    baq.forward.resize(matrix_len);
    baq.backward.resize(matrix_len);
    baq.scaling.resize(scaling_len);

//    printf("matrix index = [ ");
//    for(uint32 i = 0; i < 20; i++)
//    {
//        printf("%u, ", baq.matrix_index[i] + 0);
//    }
//    printf(" ... ");
//    for(uint32 i = baq.matrix_index.size() - 20; i < baq.matrix_index.size(); i++)
//    {
//        printf("%u, ", baq.matrix_index[i] + 0);
//    }
//    printf("]\n");
//    fflush(stdout);

    baq.reference_windows.resize(batch.device.num_reads);

    baq_state.resize(batch.device.qualities.size());
    baq.qualities.resize(batch.device.qualities.size());

    thrust::fill(baq_state.begin(), baq_state.end(), uint32(-1));
    thrust::fill(baq.qualities.begin(), baq.qualities.end(), uint8(-1));

    // compute the alignment frames
    // note: this is used both for real BAQ and flat BAQ, so we use the full active read list
    thrust::for_each(context->active_read_list.begin(),
                     context->active_read_list.end(),
                     compute_hmm_windows(*context, batch.device));

    // initialize matrices and scaling factors
    thrust::fill_n(baq.forward.begin(), baq.forward.size(), 0.0);
    thrust::fill_n(baq.backward.begin(), baq.backward.size(), 0.0);
    thrust::fill_n(baq.scaling.begin(), baq.scaling.size(), 0.0);

    // run the forward portion
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(active_baq_read_list.begin(),
                                                                  baq.matrix_index.begin(),
                                                                  baq.scaling_index.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(active_baq_read_list.end(),
                                                                  baq.matrix_index.end(),
                                                                  baq.scaling_index.end())),
                     hmm_glocal_forward(*context, batch.device, baq_state));

    // run the backward portion
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(active_baq_read_list.begin(),
                                                                  baq.matrix_index.begin(),
                                                                  baq.scaling_index.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(active_baq_read_list.end(),
                                                                  baq.matrix_index.end(),
                                                                  baq.scaling_index.end())),
                     hmm_glocal_backward(*context, batch.device, baq_state));

    // use the computed state to map qualities
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(active_baq_read_list.begin(),
                                                                  baq.matrix_index.begin(),
                                                                  baq.scaling_index.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(active_baq_read_list.end(),
                                                                  baq.matrix_index.end(),
                                                                  baq.scaling_index.end())),
                     hmm_glocal_map(*context, batch.device, baq_state));

    // for any reads that we did *not* compute a BAQ, mark the base pairs as having no BAQ uncertainty
    thrust::for_each(context->active_read_list.begin(),
                     context->active_read_list.end(),
                     read_flat_baq(*context, batch.device));

    // transform quality scores
    thrust::for_each(active_baq_read_list.begin(),
                     active_baq_read_list.end(),
                     cap_baq_qualities(*context, batch.device, baq_state));

    thrust::for_each(active_baq_read_list.begin(),
                     active_baq_read_list.end(),
                     recode_baq_qualities(*context, batch.device));

    context->stats.baq_reads += num_active;
}

void debug_baq(bqsr_context *context, const alignment_batch& batch, int read_index)
{
    const alignment_batch_host& h_batch = batch.host;

    printf("  BAQ info:\n");

    const CRQ_index idx = h_batch.crq_index(read_index);

    ushort2 read_window = context->cigar.read_window_clipped[read_index];
    uint2 reference_window = context->baq.reference_windows[read_index];

    printf("    read window                 = [ %u %u ]\n", read_window.x, read_window.y);
    printf("    absolute reference window   = [ %u %u ]\n", reference_window.x, reference_window.y);
    //printf("    sequence base: %u\n", genome.sequence_offsets[batch.alignment_sequence_IDs[read_index]]);
    printf("    relative reference window   = [ %lu %lu ]\n",
            reference_window.x - context->reference.host.sequence_bp_start[h_batch.chromosome[read_index]],
            reference_window.y - context->reference.host.sequence_bp_start[h_batch.chromosome[read_index]]);

    printf("    BAQ quals                   = [ ");
    for(uint32 i = idx.qual_start; i < idx.qual_start + idx.qual_len; i++)
    {
        uint8 q = context->baq.qualities[i];
        if (q == uint8(-1))
        {
            printf("  - ");
        } else {
            printf("% 3d ", q);
        }
    }
    printf(" ]\n");

    printf("\n");
}
