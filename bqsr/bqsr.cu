#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

#include "from_nvbio/dna.h"
#include "from_nvbio/vcf.h"

#include <map>

#include "bqsr_types.h"
#include "gamgee_loader.h"
#include "alignment_data.h"
#include "sequence_data.h"
#include "util.h"
#include "variants.h"
#include "bqsr_context.h"
#include "filters.h"
#include "cigar.h"
#include "covariates.h"
#include "baq.h"

/*
// sort batch by increasing alignment position
void device_sort_batch(BAM_alignment_batch_device *batch)
{
    D_VectorU32 temp_pos = batch->alignment_positions;

    thrust::sort_by_key(temp_pos.begin(),
                        temp_pos.begin() + temp_pos.size(),
                        batch->read_order.begin());
}
*/

void debug_read(bqsr_context *context, const alignment_batch& batch, int read_index);

void init_cuda(void)
{
    hipDeviceProp_t prop;
    int dev;
    int runtime_version;

    // trigger runtime initialization
    printf("loading CUDA runtime...\n");
    hipFree(0);

    hipRuntimeGetVersion(&runtime_version);
    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev);

    printf("CUDA runtime version: %d.%d\n", runtime_version / 1000, runtime_version % 100);
    printf("device: %s (%lu MB)\n", prop.name, prop.totalGlobalMem / (1024 * 1024));
}

int main(int argc, char **argv)
{
    // load the reference genome
    const char *ref_name = "/home/nsubtil/hg96/hs37d5.fa";
    //const char *ref_name = "/home/nsubtil/hg96/test";
    const char *vcf_name = "/home/nsubtil/hg96/ALL.chr20.integrated_phase1_v3.20101123.snps_indels_svs.genotypes-stripped.vcf";
    //const char *vcf_name = "/home/nsubtil/hg96/ALL.chr20.integrated_phase1_v3.20101123.snps_indels_svs.genotypes.vcf";
    //const char *vcf_name = "/home/nsubtil/hg96/one-variant.vcf";
    const char *bam_name = "/home/nsubtil/hg96/HG00096.chrom20.ILLUMINA.bwa.GBR.low_coverage.20120522.bam";
//    const char *bam_name = "/home/nsubtil/hg96/one-read.bam";

    init_cuda();

    sequence_data reference;

    if (gamgee_load_sequences(&reference, ref_name, SequenceDataMask::BASES | SequenceDataMask::NAMES) == false)
    {
        printf("failed to load reference %s\n", ref_name);
        exit(1);
    }

    reference.download();

    SNPDatabase_refIDs db;
    printf("loading variant database %s...\n", vcf_name);
    loadVCF(db, vcf_name);
    db.compute_sequence_offsets(reference);

    DeviceSNPDatabase dev_db;
    dev_db.load(db);


    printf("%lu variants\n", db.genome_start_positions.size());
    printf("reading BAM %s...\n", bam_name);

    gamgee_alignment_file bam(bam_name);
    alignment_batch batch;

    bqsr_context context(bam.header, dev_db, reference);

    uint32 data_mask = AlignmentDataMask::NAME |
                        AlignmentDataMask::CHROMOSOME |
                        AlignmentDataMask::ALIGNMENT_START |
                        AlignmentDataMask::CIGAR |
                        AlignmentDataMask::READS |
                        AlignmentDataMask::QUALITIES |
                        AlignmentDataMask::FLAGS |
                        AlignmentDataMask::MAPQ |
                        AlignmentDataMask::READ_GROUP;

    while(bam.next_batch(&batch, data_mask, 100000))
//    while(bam.next_batch(&batch, 500))
    {
        // load the next batch on the device
        batch.download();
        context.start_batch(batch);

        // build read offset list
        build_read_offset_list(&context, batch);
        // build read alignment window list
        build_alignment_windows(&context, batch);

        // apply read filters
        filter_reads(&context, batch);

        // apply per-BP filters
        filter_bases(&context, batch);

        // filter known SNPs from active_loc_list
        filter_known_snps(&context, batch);

        // generate cigar events and coordinates
        expand_cigars(&context, batch);

        // compute the base alignment quality for each read
        baq_reads(&context, batch);

        // build covariate tables
        gather_covariates(&context, batch);

#if 0
        for(uint32 read_id = 0; read_id < context.active_read_list.size(); read_id++)
        {
            const uint32 read_index = context.active_read_list[read_id];

            /*
            const char *name = &h_batch.names[h_batch.index[read_index].name];

            if (!strcmp(name, "SRR062635.1797528") ||
                !strcmp(name, "SRR062635.22970839") ||
                !strcmp(name, "SRR062641.22789430") ||
                !strcmp(name, "SRR062641.16264831"))
            {
                debug_read(&context, genome, h_batch, read_index);
            }*/

            debug_read(&context, batch, read_index);
        }
#endif

#if 0
        printf("active VCF ranges: %lu out of %lu reads (%f %%)\n",
                context.snp_filter.active_read_ids.size(),
                context.active_read_list.size(),
                100.0 * float(context.snp_filter.active_read_ids.size()) / context.active_read_list.size());

        H_ActiveLocationList h_bplist = context.active_location_list;
        uint32 zeros = 0;
        for(uint32 i = 0; i < h_bplist.size(); i++)
        {
            if (h_bplist[i] == 0)
                zeros++;
        }

        printf("active BPs: %u out of %u (%f %%)\n", h_bplist.size() - zeros, h_bplist.size(), 100.0 * float(h_bplist.size() - zeros) / float(h_bplist.size()));
#endif

        break;
    }

    output_covariates(&context);

    printf("%d reads filtered out of %d (%f%%)\n",
            context.stats.filtered_reads,
            context.stats.total_reads,
            float(context.stats.filtered_reads) / float(context.stats.total_reads) * 100.0);

    printf("computed base alignment quality for %d reads out of %d (%f%%)\n",
            context.stats.baq_reads,
            context.stats.total_reads - context.stats.filtered_reads,
            float(context.stats.baq_reads) / float(context.stats.total_reads - context.stats.filtered_reads) * 100.0);

    return 0;
}

void debug_read(bqsr_context *context, const alignment_batch& batch, int read_id)
{
    const alignment_batch_host& h_batch = batch.host;

    const uint32 read_index = context->active_read_list[read_id];
    const CRQ_index idx = h_batch.crq_index(read_index);

    printf("== read order %d read %d\n", read_id, read_index);

    printf("name = [%s]\n", h_batch.name[read_index].c_str());

    printf("  offset list = [ ");
    for(uint32 i = idx.read_start; i < idx.read_start + idx.read_len; i++)
    {
        uint16 off = context->read_offset_list[i];
        printf("%d ", off);
    }
    printf("]\n");

    debug_cigar(context, batch, read_index);
    debug_baq(context, batch, read_index);

    const uint2 alignment_window = context->alignment_windows[read_index];
    printf("  sequence name [%s]\n  sequence base [%lu]\n  sequence offset [%u]\n  alignment window [%u, %u]\n",
            context->reference.host.sequence_names.lookup(h_batch.chromosome[read_index]).c_str(),
            context->reference.host.sequence_bp_start[h_batch.chromosome[read_index]],
            h_batch.alignment_start[read_index],
            alignment_window.x,
            alignment_window.y);

    const uint2 vcf_range = context->snp_filter.active_vcf_ranges[read_index];
    printf("  active VCF range: [%u, %u[\n", vcf_range.x, vcf_range.y);

    printf("\n");
}
