#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

// base alignment quality calculations (gatk: BAQ.java)

#include <nvbio/basic/types.h>
#include <nvbio/basic/dna.h>
#include <nvbio/basic/primitives.h>
#include <nvbio/basic/numbers.h>

#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/functional.h>

#include <stdlib.h>
#include <math.h>

#include "bqsr_types.h"
#include "bqsr_context.h"
#include "reference.h"
#include "bam_loader.h"

#if 0
#include "baq-cpu.h"
#endif

using namespace nvbio;

#define MAX_PHRED_SCORE 93
#define EM 0.33333333333
#define EI 0.25

#define MAX_BAND_WIDTH 7
#define MIN_BASE_QUAL 4

// all bases with q < minBaseQual are up'd to this value
#define MIN_BASE_QUAL 4

#define GAP_OPEN_PROBABILITY (pow(10.0, (-40.0)/10.))
#define GAP_EXTENSION_PROBABILITY 0.1
// ola
struct compute_hmm_windows : public bqsr_lambda_ref
{
    compute_hmm_windows(bqsr_context::view ctx,
                        const reference_genome_device::const_view reference,
                        const BAM_alignment_batch_device::const_view batch)
        : bqsr_lambda_ref(ctx, reference, batch)
    { }

    NVBIO_HOST_DEVICE void operator() (const uint32 read_index)
    {
        ushort2& out_read_window = ctx.baq.read_windows[read_index];
        uint2&   out_reference_window = ctx.baq.reference_windows[read_index];

        // grab reference sequence window in the genome
        const uint32 ref_ID = batch.alignment_sequence_IDs[read_index];
        const uint32 ref_base = reference.sequence_offsets[ref_ID];
        const uint32 ref_length = reference.sequence_offsets[ref_ID + 1] - ref_base;

        const uint32 seq_to_alignment_offset = batch.alignment_positions[read_index];

        const ushort2& read_window = ctx.cigar.read_window_clipped[read_index];
        const ushort2& read_window_no_insertions = ctx.cigar.read_window_clipped_no_insertions[read_index];
        const ushort2& reference_window = ctx.cigar.reference_window_clipped[read_index];

        const uint32 first_insertion_offset = read_window_no_insertions.x - read_window.x;
        const uint32 last_insertion_offset = read_window_no_insertions.y - read_window.y;

        const int offset = MAX_BAND_WIDTH / 2;
        uint32 readStart = reference_window.x + seq_to_alignment_offset; // always clipped

        // reference window for HMM
        uint32 start = nvbio::max(readStart - offset - first_insertion_offset, 0u);
        uint32 stop = reference_window.y + seq_to_alignment_offset + offset + last_insertion_offset;

        if (stop > ref_length)
        {
            out_read_window = make_ushort2(uint16(-1), uint16(-1));
            out_reference_window = make_uint2(uint32(-1), uint32(-1));
            return;
        }

        start += ref_base;
        stop += ref_base;

        // calcBAQFromHMM line 602 starts here
        int queryStart = read_window.x;
        int queryEnd = read_window.y;

        out_read_window = make_ushort2(queryStart, queryEnd);
        out_reference_window = make_uint2(start, stop);
    }
};

// encapsulates common state for the HMM algorithm
struct hmm_common : public bqsr_lambda_ref
{
    hmm_common(bqsr_context::view ctx,
               const reference_genome_device::const_view reference,
               const BAM_alignment_batch_device::const_view batch)
        : bqsr_lambda_ref(ctx, reference, batch)
    { }

    int bandWidth, bandWidth2;

    int referenceStart, referenceLength;
    int queryStart, queryEnd, queryLen;

    double *forwardMatrix;
    double *backwardMatrix;
    double *scalingFactors;

    double sM, sI, bM, bI;

    double m[9];

    D_PackedReference referenceBases;
    D_StreamDNA16 queryBases;
    const uint8 *inputQualities;

    uint8 *outputQualities;
    uint32 *outputState;

    template<typename Tuple>
    NVBIO_HOST_DEVICE void setup(const Tuple& hmm_index)
    {
        const uint32 read_index    = thrust::get<0>(hmm_index);
        const uint32 matrix_index  = thrust::get<1>(hmm_index);
        const uint32 scaling_index = thrust::get<2>(hmm_index);

        const BAM_CRQ_index& idx = batch.crq_index[read_index];

        // set up matrix and scaling factor pointers
        forwardMatrix = &ctx.baq.forward[matrix_index];
        backwardMatrix = &ctx.baq.backward[matrix_index];
        scalingFactors = &ctx.baq.scaling[scaling_index];

        // get the windows for the current read
        const uint2& reference_window = ctx.baq.reference_windows[read_index];
        const ushort2& read_window = ctx.baq.read_windows[read_index];

        referenceStart = reference_window.x;
        referenceLength = reference_window.y - reference_window.x + 1;

        queryStart = read_window.x;
        queryEnd = read_window.y;
        queryLen = read_window.y - read_window.x + 1;

        // compute band width
        if (referenceLength > queryLen)
            bandWidth = referenceLength;
        else
            bandWidth = queryLen;

        if (MAX_BAND_WIDTH < abs(referenceLength - queryLen))
        {
            bandWidth = abs(referenceLength - queryLen) + 3;
        }

        if (bandWidth > MAX_BAND_WIDTH)
            bandWidth = MAX_BAND_WIDTH;

        if (bandWidth < abs(referenceLength - queryLen))
        {
            bandWidth = abs(referenceLength - queryLen);
        }

        bandWidth2 = bandWidth * 2 + 1;

        // initialize transition probabilities
        sM = 1.0 / (2 * queryLen + 2);
        sI = sM;
        bM = (1 - GAP_OPEN_PROBABILITY) / referenceLength;
        bI = GAP_OPEN_PROBABILITY / referenceLength;

        m[0*3+0] = (1 - GAP_OPEN_PROBABILITY - GAP_OPEN_PROBABILITY) * (1 - sM);
        m[0*3+1] = GAP_OPEN_PROBABILITY * (1 - sM);
        m[0*3+2] = m[0*3+1];
        m[1*3+0] = (1 - GAP_EXTENSION_PROBABILITY) * (1 - sI);
        m[1*3+1] = GAP_EXTENSION_PROBABILITY * (1 - sI);
        m[1*3+2] = 0.0;
        m[2*3+0] = 1 - GAP_EXTENSION_PROBABILITY;
        m[2*3+1] = 0.0;
        m[2*3+2] = GAP_EXTENSION_PROBABILITY;

//        printf("referenceStart = %u\n", referenceStart);
//        printf("queryStart = %u queryLen = %u\n", queryStart, queryLen);

        queryBases = D_StreamDNA16(batch.reads.stream(), idx.read_start + queryStart);
        referenceBases = D_PackedReference(reference.genome_stream.m_sequence_stream, referenceStart);
        inputQualities = &batch.qualities[idx.qual_start] + queryStart;

        if (ctx.baq.qualities.size() > 0)
            outputQualities = &ctx.baq.qualities[idx.qual_start] + queryStart;
        else
            outputQualities = NULL;

        if (ctx.baq.state.size() > 0)
            outputState = &ctx.baq.state[idx.qual_start] + queryStart;
        else
            outputState = NULL;

        queryStart = 0;
    }

    NVBIO_HOST_DEVICE int set_u(const int b, const int i, const int k)
    {
        int x = i - b;
        x = x > 0 ? x : 0;
        return (k + 1 - x) * 3;
    }

    // computes a matrix offset for forwardMatrix or backwardMatrix
    NVBIO_HOST_DEVICE int off(int i, int j = 0)
    {
        return i * 6 * (2 * MAX_BAND_WIDTH + 1) + j;
    }

    // computes the required HMM matrix size for the given read length
    NVBIO_HOST_DEVICE static uint32 matrix_size(const uint32 read_len)
    {
        return (read_len + 1) * 6 * (2 * MAX_BAND_WIDTH + 1);
    }

    NVBIO_HOST_DEVICE static double qual2prob(uint8 q)
    {
        return pow(10.0, -q/10.0);
    }

    NVBIO_HOST_DEVICE static double calcEpsilon(uint8 ref, uint8 read, uint8 qualB)
    {
        double qual = qual2prob(qualB < MIN_BASE_QUAL ? MIN_BASE_QUAL : qualB);
        double e = (ref == read ? 1 - qual : qual * EM);
        return e;
    }
};

struct hmm_glocal_forward : public hmm_common
{
    hmm_glocal_forward(bqsr_context::view ctx,
                       const reference_genome_device::const_view reference,
                       const BAM_alignment_batch_device::const_view batch)
        : hmm_common(ctx, reference, batch)
    { }

    template<typename Tuple>
    NVBIO_HOST_DEVICE void operator() (const Tuple& hmm_index)
    {
        const uint32 read_index = thrust::get<0>(hmm_index);

        int i, k;

        hmm_common::setup(hmm_index);

//        printf("hmm_glocal(l_ref=%d qstart=%d, l_query=%d)\n", referenceLength, queryStart, queryLen);
//        printf("ref = { ");
//        for(int c = 0; c < referenceLength; c++)
//        {
//            printf("%c ", dna_to_char(referenceBases[c]));
//        }
//        printf("\n");
//
//        printf("que = { ");
//        for(int c = 0; c < queryLen; c++)
//        {
//            printf("%c ", iupac16_to_char(queryBases[c]));
//        }
//        printf("\n");
//
//        printf("_iqual = { % 3d % 3d % 3d % 3d % 3d ... % 3d % 3d % 3d % 3d % 3d }\n",
//                inputQualities[0], inputQualities[1], inputQualities[2], inputQualities[3], inputQualities[4],
//                inputQualities[queryLen - 5], inputQualities[queryLen - 4], inputQualities[queryLen - 3], inputQualities[queryLen - 2], inputQualities[queryLen - 1]);
//        printf("c->bw = %d, bw = %d, l_ref = %d, l_query = %d\n", maxBandWidth, bandWidth, referenceLength, queryLen);

        /*** forward ***/
        // f[0]
        forwardMatrix[off(0, set_u(bandWidth, 0, 0))] = 1.0;
        scalingFactors[0] = 1.0;
        { // f[1]
            double *fi = &forwardMatrix[off(1)];
            double sum;
            int beg = 1;
            int end = referenceLength < bandWidth + 1? referenceLength : bandWidth + 1;
            int _beg, _end;

            sum = 0.0;
            for (k = beg; k <= end; ++k)
            {
                int u;
                double e = calcEpsilon(dna_to_iupac16(referenceBases[k-1]), queryBases[queryStart], inputQualities[queryStart]);
//                printf("referenceBases[%d-1] = %c inputQualities[%d] = %d queryBases[%d] = %c -> e = %.4f\n", k, dna_to_char(referenceBases[k-1]), queryStart, inputQualities[queryStart], queryStart, iupac16_to_char(queryBases[queryStart]), e);

                u = set_u(bandWidth, 1, k);

                fi[u+0] = e * bM;
                fi[u+1] = EI * bI;

                sum += fi[u] + fi[u+1];
            }

            // rescale
            scalingFactors[1] = sum;
            _beg = set_u(bandWidth, 1, beg);
            _end = set_u(bandWidth, 1, end);
            _end += 2;

            for (int k = _beg; k <= _end; ++k)
                fi[k] /= sum;
        }

        // f[2..l_query]
        for (i = 2; i <= queryLen; ++i)
        {
            double *fi = &forwardMatrix[off(i)];
            double *fi1 = &forwardMatrix[off(i-1)];
            double sum;

            int beg = 1;
            int end = referenceLength;
            int x, _beg, _end;

            char qyi = queryBases[queryStart+i-1];

            x = i - bandWidth;
            beg = beg > x? beg : x; // band start

            x = i + bandWidth;
            end = end < x? end : x; // band end

            sum = 0.0;
            for (k = beg; k <= end; ++k)
            {
                int u, v11, v01, v10;
                double e = calcEpsilon(dna_to_iupac16(referenceBases[k-1]), qyi, inputQualities[queryStart+i-1]);
//                printf("referenceBases[%d-1] = %c inputQualities[%d+%d-1] = %d qyi = %c -> e = %.4f\n", k, dna_to_char(referenceBases[k-1]), queryStart, i, inputQualities[queryStart+i-1], iupac16_to_char(qyi), e);

                u = set_u(bandWidth, i, k);
                v11 = set_u(bandWidth, i-1, k-1);
                v10 = set_u(bandWidth, i-1, k);
                v01 = set_u(bandWidth, i, k-1);

                fi[u+0] = e * (m[0] * fi1[v11+0] + m[3] * fi1[v11+1] + m[6] * fi1[v11+2]);
                fi[u+1] = EI * (m[1] * fi1[v10+0] + m[4] * fi1[v10+1]);
                fi[u+2] = m[2] * fi[v01+0] + m[8] * fi[v01+2];

                sum += fi[u] + fi[u+1] + fi[u+2];

    //            printf("(%d,%d;%d): %.4f,%.4f,%.4f\n", i, k, u, fi[u], fi[u+1], fi[u+2]);
    //            printf(" .. u = %d v11 = %d v01 = %d v10 = %d e = %f\n", u, v11, v01, v10, e);
            }

            // rescale
            scalingFactors[i] = sum;

            _beg = set_u(bandWidth, i, beg);
            _end = set_u(bandWidth, i, end);
            _end += 2;

            for (k = _beg, sum = 1./sum; k <= _end; ++k)
                fi[k] *= sum;
        }

        { // f[l_query+1]
            double sum = 0.0;

            for (k = 1; k <= referenceLength; ++k)
            {
                int u = set_u(bandWidth, queryLen, k);

                if (u < 3 || u >= bandWidth2*3+3)
                    continue;

                sum += forwardMatrix[off(queryLen,u+0)] * sM + forwardMatrix[off(queryLen, u+1)] * sI;
            }

            scalingFactors[queryLen+1] = sum; // the last scaling factor
        }
    }
};

struct hmm_glocal_backward : public hmm_common
{
    hmm_glocal_backward(bqsr_context::view ctx,
                        const reference_genome_device::const_view reference,
                        const BAM_alignment_batch_device::const_view batch)
        : hmm_common(ctx, reference, batch)
    { }

    template<typename Tuple>
    NVBIO_HOST_DEVICE void operator() (const Tuple& hmm_index)
    {
        const uint32 read_index = thrust::get<0>(hmm_index);

        int i, k;

        hmm_common::setup(hmm_index);

        /*** backward ***/
        // b[l_query] (b[l_query+1][0]=1 and thus \tilde{b}[][]=1/s[l_query+1]; this is where s[l_query+1] comes from)
        for (k = 1; k <= referenceLength; ++k)
        {
            int u = set_u(bandWidth, queryLen, k);
            double *bi = &backwardMatrix[off(queryLen)];

            if (u < 3 || u >= bandWidth2*3+3)
                continue;

            bi[u+0] = sM / scalingFactors[queryLen] / scalingFactors[queryLen+1];
            bi[u+1] = sI / scalingFactors[queryLen] / scalingFactors[queryLen+1];
        }

        // b[l_query-1..1]
        for (i = queryLen - 1; i >= 1; --i)
        {
            int beg = 1;
            int end = referenceLength;
            int x, _beg, _end;

            double *bi = &backwardMatrix[off(i)];
            double *bi1 = &backwardMatrix[off(i+1)];
            double y = (i > 1)? 1. : 0.;

            char qyi1 = queryBases[queryStart+i];

            x = i - bandWidth;
            beg = beg > x? beg : x;

            x = i + bandWidth;
            end = end < x? end : x;

            for (k = end; k >= beg; --k)
            {
                int u, v11, v01, v10;

                u = set_u(bandWidth, i, k);
                v11 = set_u(bandWidth, i+1, k+1);
                v10 = set_u(bandWidth, i+1, k);
                v01 = set_u(bandWidth, i, k+1);

                /* const */ double e;
                if (k >= referenceLength)
                    e = 0;
                else
                    e = calcEpsilon(dna_to_iupac16(referenceBases[k]), qyi1, inputQualities[queryStart+i]) * bi1[v11];

                bi[u+0] = e * m[0] + EI * m[1] * bi1[v10+1] + m[2] * bi[v01+2]; // bi1[v11] has been folded into e.
                bi[u+1] = e * m[3] + EI * m[4] * bi1[v10+1];
                bi[u+2] = (e * m[6] + m[8] * bi[v01+2]) * y;
            }

            // rescale
            _beg = set_u(bandWidth, i, beg);
            _end = set_u(bandWidth, i, end);
            _end += 2;

            y = 1.0 / scalingFactors[i];
            for (k = _beg; k <= _end; ++k)
                bi[k] *= y;
        }

//        double pb = 0.0;
        { // b[0]
            int beg = 1;
            int end = referenceLength < bandWidth + 1? referenceLength : bandWidth + 1;

            double sum = 0.0;
            for (k = end; k >= beg; --k)
            {
                int u = set_u(bandWidth, 1, k);
                double e = calcEpsilon(dna_to_iupac16(referenceBases[k-1]), queryBases[queryStart], inputQualities[queryStart]);

                if (u < 3 || u >= bandWidth2*3+3)
                    continue;

                sum += e * backwardMatrix[off(1, u+0)] * bM + EI * backwardMatrix[off(1, u+1)] * bI;
            }

            backwardMatrix[off(0, set_u(bandWidth, 0, 0))] = sum / scalingFactors[0];
//            pb = backwardMatrix[off(0, set_u(bandWidth, 0, 0))]; // if everything works as is expected, pb == 1.0
        }
    }
};

struct hmm_glocal_map : public hmm_common
{
    hmm_glocal_map(bqsr_context::view ctx,
                   const reference_genome_device::const_view reference,
                   const BAM_alignment_batch_device::const_view batch)
        : hmm_common(ctx, reference, batch)
    { }

    template<typename Tuple>
    NVBIO_HOST_DEVICE void operator() (const Tuple& hmm_index)
    {
        const uint32 read_index = thrust::get<0>(hmm_index);

        int i, k;

        hmm_common::setup(hmm_index);

        /*** MAP ***/
        for (i = 1; i <= queryLen; ++i)
        {
            double sum = 0.0;
            double max = 0.0;

            const double *fi = &forwardMatrix[off(i)];
            const double *bi = &backwardMatrix[off(i)];

            int beg = 1;
            int end = referenceLength;
            int x, max_k = -1;

            x = i - bandWidth;
            beg = beg > x? beg : x;

            x = i + bandWidth;
            end = end < x? end : x;

            for (k = beg; k <= end; ++k)
            {
                const int u = set_u(bandWidth, i, k);
                double z = 0.0;

                z = fi[u+0] * bi[u+0];
                sum += z;
                if (z > max)
                {
                    max = z;
                    max_k = (k-1) << 2 | 0;
                }

                z = fi[u+1] * bi[u+1];
                sum += z;
                if (z > max)
                {
                    max = z;
                    max_k = (k-1) << 2 | 1;
                }
            }

            max /= sum;
            sum *= scalingFactors[i]; // if everything works as is expected, sum == 1.0

            if (outputState != NULL)
                outputState[queryStart+i-1] = max_k;

            if (outputQualities != NULL)
            {
                k = (int)(double(-4.343) * log(double(1.0) - double(max)) + double(.499)); // = 10*log10(1-max)
                outputQualities[queryStart+i-1] = (char)(k > 100? 99 : (k < MIN_BASE_QUAL ? MIN_BASE_QUAL : k));

    //            printf("outputQualities[%d]: max = %.16f l = %.4f dk = %.4f k = %d -> %d\n", i, max, l, dk, k, outputQualities[queryStart+i-1]);
            }

    //        printf("(%.4f,%.4f) (%d,%d,%d,%.4f)\n", pb, sum, (i-1), (max_k>>2), (max_k&3), max);
        }
    }
};

// functor to compute the size required for the forward/backward HMM matrix
// note that this computes the size required for *one* matrix only; we allocate the matrices on two separate vectors and use the same index for both
struct compute_hmm_matrix_size : public thrust::unary_function<uint32, uint32>, public bqsr_lambda
{
    compute_hmm_matrix_size(bqsr_context::view ctx,
                            const BAM_alignment_batch_device::const_view batch)
        : bqsr_lambda(ctx, batch)
    { }

    NVBIO_HOST_DEVICE uint32 operator() (const uint32 read_index)
    {
        const BAM_CRQ_index& idx = batch.crq_index[read_index];
        return hmm_common::matrix_size(idx.read_len);
    }
};

struct compute_hmm_scaling_factor_size : public thrust::unary_function<uint32, uint32>, public bqsr_lambda
{
    compute_hmm_scaling_factor_size(bqsr_context::view ctx,
                                    const BAM_alignment_batch_device::const_view batch)
        : bqsr_lambda(ctx, batch)
    { }

    NVBIO_HOST_DEVICE uint32 operator() (const uint32 read_index)
    {
        const BAM_CRQ_index& idx = batch.crq_index[read_index];
        return idx.read_len + 2;
    }
};

void baq_reads(bqsr_context *context, const reference_genome& reference, const BAM_alignment_batch_device& batch)
{
    struct baq_context& baq = context->baq;

    // compute the index and size of the HMM matrices
    baq.matrix_index.resize(context->active_read_list.size() + 1);
    // first offset is zero
    thrust::fill_n(baq.matrix_index.begin(), 1, 0);
    // do an inclusive scan to compute all offsets + the total size
    nvbio::inclusive_scan(context->active_read_list.size(),
                          thrust::make_transform_iterator(context->active_read_list.begin(),
                                                          compute_hmm_matrix_size(*context, batch)),
                          baq.matrix_index.begin() + 1,
                          thrust::plus<uint32>(),
                          context->temp_storage);

    // compute the index and size of the HMM scaling factors
    baq.scaling_index.resize(context->active_read_list.size() + 1);
    // first offset is zero
    thrust::fill_n(baq.scaling_index.begin(), 1, 0);
    nvbio::inclusive_scan(context->active_read_list.size(),
                          thrust::make_transform_iterator(context->active_read_list.begin(),
                                                          compute_hmm_scaling_factor_size(*context, batch)),
                          baq.scaling_index.begin() + 1,
                          thrust::plus<uint32>(),
                          context->temp_storage);

    // read back the last elements, which contain the size of the buffer required
    uint32 matrix_len = baq.matrix_index[context->active_read_list.size()];
    uint32 scaling_len = baq.scaling_index[context->active_read_list.size()];

//    printf("reads: %u\n", batch.num_reads);
//    printf("forward len = %u bytes = %lu\n", matrix_len, matrix_len * sizeof(double));
//    printf("expected len = %lu expected bytes = %lu\n",
//            hmm_common::matrix_size(100) * context->active_read_list.size(),
//            hmm_common::matrix_size(100) * context->active_read_list.size() * sizeof(double));
//    printf("per read matrix size = %u bytes = %lu\n", hmm_common::matrix_size(100), hmm_common::matrix_size(100) * sizeof(double));

    baq.forward.resize(matrix_len);
    baq.backward.resize(matrix_len);
    baq.scaling.resize(scaling_len);

//    printf("matrix index = [ ");
//    for(uint32 i = 0; i < 20; i++)
//    {
//        printf("%u, ", baq.matrix_index[i] + 0);
//    }
//    printf(" ... ");
//    for(uint32 i = baq.matrix_index.size() - 20; i < baq.matrix_index.size(); i++)
//    {
//        printf("%u, ", baq.matrix_index[i] + 0);
//    }
//    printf("]\n");
//    fflush(stdout);

    baq.read_windows.resize(batch.num_reads);
    baq.reference_windows.resize(batch.num_reads);

    baq.state.resize(batch.qualities.size());
    baq.qualities.resize(batch.qualities.size());

    thrust::fill(baq.state.begin(), baq.state.end(), uint32(-1));
    thrust::fill(baq.qualities.begin(), baq.qualities.end(), uint8(-1));

    // compute the alignment frames
    thrust::for_each(context->active_read_list.begin(),
                     context->active_read_list.end(),
                     compute_hmm_windows(*context, reference.device, batch));

    // initialize matrices and scaling factors
    thrust::fill_n(baq.forward.begin(), baq.forward.size(), 0.0);
    thrust::fill_n(baq.backward.begin(), baq.backward.size(), 0.0);
    thrust::fill_n(baq.scaling.begin(), baq.scaling.size(), 0.0);

    // run the forward portion
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(context->active_read_list.begin(),
                                                                  baq.matrix_index.begin(),
                                                                  baq.scaling_index.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(context->active_read_list.end(),
                                                                  baq.matrix_index.end(),
                                                                  baq.scaling_index.end())),
                     hmm_glocal_forward(*context, reference.device, batch));

    // run the backward portion
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(context->active_read_list.begin(),
                                                                  baq.matrix_index.begin(),
                                                                  baq.scaling_index.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(context->active_read_list.end(),
                                                                  baq.matrix_index.end(),
                                                                  baq.scaling_index.end())),
                     hmm_glocal_backward(*context, reference.device, batch));

    // use the computed state to map qualities
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(context->active_read_list.begin(),
                                                                  baq.matrix_index.begin(),
                                                                  baq.scaling_index.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(context->active_read_list.end(),
                                                                  baq.matrix_index.end(),
                                                                  baq.scaling_index.end())),
                     hmm_glocal_map(*context, reference.device, batch));

#if 0
    for(uint32 i = 0; i < context->active_read_list.size(); i++)
    {
        uint32 read_index = context->active_read_list[i];

        ushort2 read_window;
        uint2 reference_window;

        cpu::frame_alignment(read_window, reference_window,
                             context, reference, batch, h_batch, read_index);
        //hengli::calcBAQFromHMM(NULL, NULL, NULL, context, reference, h_batch, read_index);
        printf("cpu frame_alignment: read = [ %u %u ] ref = [ %u %u ]\n", read_window.x, read_window.y, reference_window.x, reference_window.y);

        read_window = baq.read_windows[read_index];
        reference_window = baq.reference_windows[read_index];
        printf("gpu frame_alignment: read = [ %u %u ] ref = [ %u %u ]\n", read_window.x, read_window.y, reference_window.x, reference_window.y);
    }
#endif
}

void debug_baq(bqsr_context *context, const reference_genome& genome, const BAM_alignment_batch_host& batch, int read_index)
{
    printf("  BAQ info:\n");

    const BAM_CRQ_index& idx = batch.crq_index[read_index];

    ushort2 read_window = context->baq.read_windows[read_index];
    uint2 reference_window = context->baq.reference_windows[read_index];

    printf("    read window                 = [ %u %u ]\n", read_window.x, read_window.y);
    printf("    absolute reference window   = [ %u %u ]\n", reference_window.x, reference_window.y);
    //printf("    sequence base: %u\n", genome.sequence_offsets[batch.alignment_sequence_IDs[read_index]]);
    printf("    relative reference window   = [ %u %u ]\n",
            reference_window.x - genome.sequence_offsets[batch.alignment_sequence_IDs[read_index]],
            reference_window.y - genome.sequence_offsets[batch.alignment_sequence_IDs[read_index]]);

    printf("    BAQ state                   = [ ");
    for(uint32 i = 0; i < idx.read_len; i++)
    {
        uint32 s = context->baq.state[i];

        if (s == uint32(-1))
        {
            printf("  - ");
        } else {
            printf("% 3d ", s);
        }
    }
    printf(" ]\n");

    printf("    BAQ quals                   = [ ");
    for(uint32 i = 0; i < idx.read_len; i++)
    {
        uint8 q = context->baq.qualities[i];
        if (q == uint8(-1))
        {
            printf("  - ");
        } else {
            printf("% 3d ", q);
        }
    }
    printf(" ]\n");

    printf("\n");
}
