#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

#include <nvbio/basic/types.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/dna.h>
#include <nvbio/io/sequence/sequence.h>
#include <nvbio/io/sequence/sequence_access.h>
#include <nvbio/io/vcf.h>
#include <nvbio/io/sequence/sequence_pac.h>

#include <map>

#include "bam_loader.h"
#include "reference.h"
#include "util.h"
#include "variants.h"
#include "bqsr_context.h"
#include "filters.h"
#include "cigar.h"
#include "covariates.h"
#include "baq.h"

using namespace nvbio;

/*
// sort batch by increasing alignment position
void device_sort_batch(BAM_alignment_batch_device *batch)
{
    D_VectorU32 temp_pos = batch->alignment_positions;

    thrust::sort_by_key(temp_pos.begin(),
                        temp_pos.begin() + temp_pos.size(),
                        batch->read_order.begin());
}
*/

void debug_read(bqsr_context *context, const reference_genome& reference, const BAM_alignment_batch_host& batch, int read_index);

int main(int argc, char **argv)
{
    // load the reference genome
    const char *ref_name = "hs37d5";
    //const char *ref_name = "/home/nsubtil/hg96/test";
    const char *vcf_name = "/home/nsubtil/hg96/ALL.chr20.integrated_phase1_v3.20101123.snps_indels_svs.genotypes-stripped.vcf";
    //const char *vcf_name = "/home/nsubtil/hg96/ALL.chr20.integrated_phase1_v3.20101123.snps_indels_svs.genotypes.vcf";
    //const char *vcf_name = "/home/nsubtil/hg96/one-variant.vcf";
    const char *bam_name = "/home/nsubtil/hg96/HG00096.chrom20.ILLUMINA.bwa.GBR.low_coverage.20120522.bam";
    //const char *bam_name = "/home/nsubtil/hg96/one-read.bam";

    struct reference_genome reference;

    printf("loading reference %s...\n", ref_name);

    if (reference.load(ref_name) == false)
    {
        printf("failed to load reference %s\n", ref_name);
        exit(1);
    }

    reference.download();

    SNPDatabase_refIDs db;
    printf("loading variant database %s...\n", vcf_name);
    io::loadVCF(db, vcf_name);
    db.compute_sequence_offsets(reference);

    DeviceSNPDatabase dev_db;
    dev_db.load(db);


    printf("%lu variants\n", db.genome_start_positions.size());
    printf("reading BAM %s...\n", bam_name);

    BAMfile bam(bam_name);

    BAM_alignment_batch batch;

    bqsr_context context(bam.header, dev_db, reference);

    while(bam.next_batch(&batch, true, 100000))
//    while(bam.next_batch(&batch, false, 500))
    {
        // load the next batch on the device
        batch.download();
        context.start_batch(batch);

        // build read offset list
        build_read_offset_list(&context, batch);
        // build read alignment window list
        build_alignment_windows(&context, batch);

        // apply read filters
        filter_reads(&context, batch);

        // apply per-BP filters
        filter_bases(&context, batch);

        // filter known SNPs from active_loc_list
        filter_known_snps(&context, batch);

        // generate cigar events and coordinates
        expand_cigars(&context, batch);

        // compute the base alignment quality for each read
        baq_reads(&context, batch);

        // build covariate tables
        gather_covariates(&context, batch);

#if 0
        for(uint32 read_id = 0; read_id < context.active_read_list.size(); read_id++)
        {
            const uint32 read_index = context.active_read_list[read_id];

            /*
            const char *name = &h_batch.names[h_batch.index[read_index].name];

            if (!strcmp(name, "SRR062635.1797528") ||
                !strcmp(name, "SRR062635.22970839") ||
                !strcmp(name, "SRR062641.22789430") ||
                !strcmp(name, "SRR062641.16264831"))
            {
                debug_read(&context, genome, h_batch, read_index);
            }*/

            debug_read(&context, reference, h_batch, read_index);
        }
#endif

#if 0
        printf("active VCF ranges: %lu out of %lu reads (%f %%)\n",
                context.snp_filter.active_read_ids.size(),
                context.active_read_list.size(),
                100.0 * float(context.snp_filter.active_read_ids.size()) / context.active_read_list.size());

        H_ActiveLocationList h_bplist = context.active_location_list;
        uint32 zeros = 0;
        for(uint32 i = 0; i < h_bplist.size(); i++)
        {
            if (h_bplist[i] == 0)
                zeros++;
        }

        printf("active BPs: %u out of %u (%f %%)\n", h_bplist.size() - zeros, h_bplist.size(), 100.0 * float(h_bplist.size() - zeros) / float(h_bplist.size()));
#endif

        break;
    }

    printf("%d reads filtered out of %d (%f%%)\n",
            context.stats.filtered_reads,
            context.stats.total_reads,
            float(context.stats.filtered_reads) / float(context.stats.total_reads) * 100.0);

    printf("computed base alignment quality for %d reads out of %d (%f%%)\n",
            context.stats.baq_reads,
            context.stats.total_reads - context.stats.filtered_reads,
            float(context.stats.baq_reads) / float(context.stats.total_reads - context.stats.filtered_reads) * 100.0);

    return 0;
}

void debug_read(bqsr_context *context, const BAM_alignment_batch& batch, int read_id)
{
    const BAM_alignment_batch_host& h_batch = batch.host;

    uint32 read_index = context->active_read_list[read_id];

    io::SequenceDataView view = plain_view(*(context->reference.h_ref));
    H_PackedReference reference_stream(view.m_sequence_stream);
    const BAM_CRQ_index& idx = h_batch.crq_index[read_index];

    printf("== read order %d read %d\n", read_id, read_index);

    printf("name = [%s]\n", &h_batch.names[h_batch.index[read_index].name]);

    printf("  offset list = [ ");
    for(uint32 i = idx.read_start; i < idx.read_start + idx.read_len; i++)
    {
        uint16 off = context->read_offset_list[i];
        printf("%d ", off);
    }
    printf("]\n");

    debug_cigar(context, batch, read_index);
    debug_baq(context, batch, read_index);

    const uint2 alignment_window = context->alignment_windows[read_index];
    printf("  sequence name [%s]\n  sequence base [%u]\n  sequence offset [%u]\n  alignment window [%u, %u]\n",
            &view.m_name_stream[view.m_name_index[h_batch.alignment_sequence_IDs[read_index]]],
            context->reference.sequence_offsets[h_batch.alignment_sequence_IDs[read_index]],
            h_batch.alignment_positions[read_index],
            alignment_window.x,
            alignment_window.y);

    const uint2 vcf_range = context->snp_filter.active_vcf_ranges[read_index];
    printf("  active VCF range: [%u, %u[\n", vcf_range.x, vcf_range.y);

    printf("\n");
}
